﻿char *title = "Floyd's algorithm";
char *description = "Алгоритм Флойда - поиск всех кратчайших путей в графе";
/*
Алгоритм Флойда является одним из методов поиска кратчайших путей в графе. 
В отличии от алгоритма Дейкстры, который позволяет при доведении до конца построить 
ориентированное дерево кратчайших путей от некоторой вершины, метод Флойда позволяет 
найти длины всех кратчайших путей в графе. Конечно эта задача может быть решена 
и многократным применением алгоритма Дейкстры (каждый раз последовательно выбираем 
вершину от первой до N-ной, пока не получим кратчайшие пути от всех вершин графа), 
однако реализация подобной процедуры потребовала бы значительных вычислительных затрат.

Прежде чем представлять алгоритмы, необходимо ввести некоторые обозначения. 
Перенумеруем вершины исходного графа целыми числами от 1 до N. Обозначим через di,jm длину кратчайшего пути 
из вершинм i в вершину j, который в качестве промежуточных может содержать только первые m вершин графа. 
(Напомним, что промежуточной вершиной пути является любая принадлежащая ему вершина, не совпадающая 
с его начальной или конечной вершинами.) Если между вершинами i и j не существует ни одного пути указанного типа, 
то условно будем считать, что di,jm=∞. Из данного определения величин di,jm следует, что величина di,j0, 
представляет длину кратчайшего пути из вершины i в вершину j, не имеющего промежуточных вершин, 
т. е. длину кратчайшей дуги, соединяющей i с j (если такие дуги присутствуют в графе). 
для любой вершины i положим di,im= 0. Отметим далее, что величина di,jmпредставляет длину кратчайшего пути 
между вершинами i и j.

Обозначим через Dm матрицу размера NxN, элемент (i, j) которой совпадает с di,jm. 
Если в исходном графе нам известна длина каждой дуги, то мы можем сформировать матрицу D0. 
Наша цель состоит в определении матрицы DN, представляющей кратчайшие пути между всеми вершинами рассматриваемого графа.

В алгоритме Флойда в качестве исходной выступает матрица D0. 
Вначале из этой матрицы вычисляется матрица D1. 
Затем по матрице D1 вычисляется матрицав D2 и т. д. 
Процесс повторяется до тех пор, пока по матрице DN-1 не будет вычислена матрица DN.

Рассмотрим основную идею, лежащую в основе алгоритма Флойда. 
Суть алгоритма Флойда заключается в проверке того, не окажется ли путь из вершины i в вершину j короче, 
если он будет проходить через некоторую промежуточную вершину m. Предположим, что нам известны:

кратчайший путь из вершины i в вершину m, в котором в качестве промежуточных допускается использование только первых (m - 1) вершин;
кратчайший путь из вершины m в вершину j, в котором в качестве промежуточных допускается использование только первых (m - 1) вершин;
кратчайший путь из вершины i в вершину j, в котором в качестве промежуточных допускается использование только первых (m - 1) вершин.

Поскольку по предположению исходный граф не может содержать контуров отрицательной длины, 
один из двух путей — путь, совпадающий с представленным в пункте 3, или путь, являющийся объединением 
путей из пунктов 1 и 2 — должен быть кратчайшим путем из вершины i в вершину j, 
в котором в качестве промежуточных допускается использование только первых m вершин. Таким образом,

di,jm=min{ di,mm-1+ dm,jm-1; di,jm-1}

Из соотношения видно, что для вычисления элементов матрицы Dm необходимо располагать лишь элементами матрицы Dm-1. 
Более того, соответствующие вычисления могут быть проведены без обращения к исходному графу. 
Теперь мм в состоянии дать формальное описание алгоритма Флойда для нахождения на графе кратчайших путей 
между всеми парами вершин. 

Алгоритм

Перенумеровать вершины графа от 1 до N целыми числами, определить матрицу D0, каждый элемент di,j  
которой есть длина кратчайшей дуги между вершинами i и j. Если такой дуги нет, положить значение элемента 
равным ∞. Кроме того, положить значения диагонального элемента di,iравным 0.
Для целого m, последовательно принимающего значения 1...N определить по элементам матрицы Dm-1 элементы Dm
Алгоритм заканчивается получением матрицы всех кратчайших путей DN, N – число вершин графа.
 
Напомним, для определения по известным элементам матрицы Dm-1 элементов матрицы  Dm 
в алгоритме Флойда применяется рекурсивное соотношение:

di,jm=min{ di,mm-1+ dm,jm-1; di,jm-1}

di,jm – элемент матрицы Dm, di,jm-1 – элементы матрицы Dm-1 найденой на предыдущем шаге алгоритма.
*/
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <ctype.h>
#include <limits.h>
#include <hip/hip_runtime.h>



#define assert( bool ) 
int strempty(const char *p)
{
	if (!p)
		return (1);
	for (; *p; p++)
		if (!isspace(*p))
			return (0);
	return (1);
}
char *mystrtok(char **m, char *s, char c)
{
	char *p = s ? s : *m;
	if (!*p)
		return 0;
	*m = strchr(p, c);
	if (*m)
		*(*m)++ = 0;
	else
		*m = p + strlen(p);
	return p;
}
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

__global__ void global_init(long *prev, long *next, int *intermedian, int m, int n){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		intermedian[id] = -1;
	}
}

__global__ void global_floyd(long *prev, long *next, int *intermedian, int m, int n){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		int i = id/n;
		int j = id%n;
		if(i != j && prev[i*n+m] != LONG_MAX && prev[m*n+j] != LONG_MAX && (prev[i*n+m] + prev[m*n+j]) < prev[i*n+j]){
			next[id] = prev[i*n+m] + prev[m*n+j];
			intermedian[id] = m;
		}
		else {
			next[id] = prev[id];
		}
	}
}

__host__ void host_floyd(int gridSize, int blockSize, long *matrix, int *intermedian, int n)
{
	hipError_t err;
	long *device_prev;
	long *device_next;
	int *device_intermedian;

	err = hipMalloc((void**)&device_prev, n*n * sizeof(long));
	err = hipMalloc((void**)&device_next, n*n * sizeof(long));
	err = hipMalloc((void**)&device_intermedian, n*n * sizeof(int));

	err = hipMemcpy(device_prev, matrix, n*n*sizeof(long), hipMemcpyHostToDevice);

	int blocks = (gridSize > 0)? gridSize : min(max(1, (int)pow((double)n*n, 0.333333333333333)), 15);
	int threads = (blockSize > 0)? blockSize : min(max(1, (int)pow((double)n*n, 0.333333333333333)), 15);
	global_init <<< blocks, threads >>>(device_prev, device_next, device_intermedian, -1, n);

	for(int m = 0; m < n ; m++){
		global_floyd <<< blocks, threads >>>(device_prev, device_next, device_intermedian, m, n);
		err = hipMemcpy(matrix, device_next, n*n*sizeof(long), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++){
			for (int j = 0; j < n; j++){
				printf("%ld%s", matrix[i*n + j], ((j == n - 1) ? "\n" : ";"));
			}
		}
		printf("\n");
		long * t = device_prev; device_prev = device_next; device_next = t;
	}

	err = hipMemcpy(matrix, device_prev, n*n*sizeof(long), hipMemcpyDeviceToHost);
	err = hipMemcpy(intermedian, device_intermedian, n*n*sizeof(int), hipMemcpyDeviceToHost);

	err = hipFree(device_prev);
	err = hipFree(device_next);
	err = hipFree(device_intermedian);

	err = err;
}

int main(int argc, char* argv[])
{
	int gridSize = 0;
	int blockSize = 0;

	printf("Title :\t%s\n", title); fflush(stdout);

	if (argc < 4) {
		printf("Usage :\t%s [-g <gridSize>] [-b <blockSize>] <inputfilename> <outputfilename> <intermedianfilename>\n", argv[0]); fflush(stdout);
		printf("\tinputfilename - source matrix of path prices or empty\n"); fflush(stdout);
		printf("\toutputfilename - output floyd's matrix of path prices\n"); fflush(stdout);
		printf("\tintermedianfilename - output matrix of intermedian points or empty\n"); fflush(stdout);
		exit(-1);
	}

	int argId = 1;
	for(; argId < argc && argv[argId][0]=='-' ; argId++){
		switch(argv[argId][1]){
		case 'g':
			gridSize = atoi(argv[++argId]);
			break;
		case 'b':
			blockSize = atoi(argv[++argId]);
			break;
		}
	}

	char *inputFileName = argv[argId++];
	char *outputFileName = argv[argId++];
	char *intermedianFileName = argv[argId++];

	printf("Input File Name :\t%s\n", inputFileName); fflush(stdout);
	printf("Output File Name :\t%s\n", outputFileName); fflush(stdout);
	printf("Intermedian File Name :\t%s\n", intermedianFileName); fflush(stdout);

	char buffer[4096];
	char *tok;
	char *p;
	int n;         /* Ранг текущего массива */
	long *matrix;  /* Массив цен */
	int *intermedian;  /* Массивов промежуточных точек */
	int i, j;

	FILE *fs = fopen(inputFileName, "r");
	if (fs == NULL) {
		fprintf(stderr, "File open error (%s)\n", inputFileName); fflush(stderr);
		exit(-1);
	}

	n = 0;

	/* Заполняем массив числами из файла */
	/* Операция выполняетя только на хост процессе */
	/* Операция выполняетя в два прохода по файлу */
	/* На первом проходе определяется ранг матрицы */
	/* На втором проходе считываются данные */
	for (i = 0; (tok = fgets(buffer, sizeof(buffer), fs)) != NULL; i++)
	{
		j = 0;
		for (tok = mystrtok(&p, tok, ';'); tok != NULL; tok = mystrtok(&p, NULL, ';'))
		{
			j++;
		}
		n = max(n, j);
	}
	n = max(n, i);

	matrix = (long *)malloc(n*n*sizeof(long));
	intermedian = (int *)malloc(n*n*sizeof(int));

	fseek(fs, 0, SEEK_SET);

	for (i = 0; (tok = fgets(buffer, sizeof(buffer), fs)) != NULL; i++)
	{
		j = 0;
		for (tok = mystrtok(&p, tok, ';'); tok != NULL; tok = mystrtok(&p, NULL, ';'))
		{
			/* Пустые элементы - это запрещённые пути */
			matrix[n*i + j++] = strempty(tok) ? LONG_MAX : atol(tok);
		}
		for (; j < n; j++) matrix[n*i + j] = LONG_MAX;
	}
	for (j = 0; j < (n - i)*n; j++) matrix[n*i + j] = LONG_MAX;
	for (i = 0; i < n; i++) matrix[n*i + i] = LONG_MAX; /* Запрещаем петли */

	fclose(fs);

	printf("Matrix rank :\t%d\n", n);
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			printf("%ld%s", matrix[i*n + j], ((j == n - 1) ? "\n" : "\t"));
		}
	}
	fflush(stdout);

	// Find/set the device.
	int device_size = 0;
	hipGetDeviceCount(&device_size);
	for (i = 0; i < device_size; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		printf("Running on GPU %d (%s)\n", i, properties.name); fflush(stdout);
	}

	host_floyd(gridSize, blockSize, matrix, intermedian, n);

	hipDeviceReset();

	/* Bыводим результаты */
	fs = fopen(outputFileName, "w");
	if (fs == NULL) {
		fprintf(stderr, "File open error (%s)\n", outputFileName); fflush(stderr);
		exit(-1);
	}
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			if (matrix[i*n + j] != LONG_MAX)
				fprintf(fs, "%ld%s", matrix[i*n + j], ((j == n - 1) ? "\n" : ";"));
			else
				fprintf(fs, "%s", ((j == n - 1) ? "\n" : ";"));
		}
	}
	fclose(fs);

	/* Bыводим результаты */
	fs = fopen(intermedianFileName, "w");
	if (fs == NULL) {
		fprintf(stderr, "File open error (%s)\n", intermedianFileName); fflush(stderr);
		exit(-1);
	}
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			if (intermedian[i*n + j] >= 0)
				fprintf(fs, "%d%s", intermedian[i*n + j], ((j == n - 1) ? "\n" : ";"));
			else
				fprintf(fs, "%s", ((j == n - 1) ? "\n" : ";"));
		}
	}
	fclose(fs);

	free(matrix);
	free(intermedian);

	exit(0);
}