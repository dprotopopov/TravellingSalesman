#include "hip/hip_runtime.h"
﻿char *title = "Little's algorithm";
char *description = "Алгоритм Литтла - метод решения задачи коммивояжера";
/*
Алгоритм Литтла применяют для поиска решения задачи коммивояжера в виде гамильтонова контура.
Данный алгоритм используется для поиска оптимального гамильтонова контура в графе, имеющем N вершин,
причем каждая вершина i связана с любой другой вершиной j двунаправленной дугой.
Каждой дуге приписан вес Сi,j, причем веса дуг строго положительны (Сi,j≥0).
Веса дуг образуют матрицу стоимости. Все элементы по диагонали матрицы приравнивают
к бесконечности (Сj,j=∞).

В случае, если пара вершин i и j не связана между собой (граф не полносвязный), то соответствующему элементу
матрицы стоимости приписываем вес, равный длине минимального пути между вершинами i и j.
Если в итоге дуга (i, j) войдет в результирующий контур, то ее необходимо заменить соответствующим ей путем.
Матрицу оптимальных путей между всеми вершинами графа можно получить применив алгоритм Данцига или Флойда.

Алгоритм Литтала является частным случаем применения метода "ветвей и границ" для конкретной задачи.
Общая идея тривиальна: нужно разделить огромное число перебираемых вариантов на классы и получить оценки
(снизу – в задаче минимизации, сверху – в задаче максимизации) для этих классов, чтобы иметь возможность
отбрасывать варианты не по одному, а целыми классами.
Трудность состоит в том, чтобы найти такое разделение на классы (ветви) и такие оценки (границы),
чтобы процедура была эффективной.

Алгоритм Литтла

В каждой строке матрицы стоимости найдем минимальный элемент и вычтем его из всех элементов строки.
Сделаем это и для столбцов.
Получим матрицу стоимости, каждая строка и каждый столбец которой содержат хотя бы один нулевой элемент.
Для каждого нулевого элемента матрицы cij  рассчитаем коэффициент Гi,j, который равен сумме наименьшего элемента i строки
(исключая элемент Сi,j=0) и наименьшего элемента j столбца.
Проверяем, что не существует однозначных путей - то есть с одним входом и выходом
Если такой путь есть, то выбираем его
иначе Из всех коэффициентов  Гi,j выберем такой, который является максимальным Гk,m=max{Гi,j}.
В гамильтонов контур вносится соответствующая дуга (k,m).
Удаляем k-тую строку и столбец m. 
Поменяем на бесконечность значение элемент Сr,l для всех путей (l,...,k,m,...r) из добавленных дуг, 
содежащих дугу (k,m) (иначе может образоваться простой цикл).
Повторяем алгоритм шага 1, пока порядок матрицы не станет равным одному.
Получаем гамильтонов контур.
В ходе решения ведется постоянный подсчет текущего значения нижней границы.
Нижняя граница равна сумме всех вычтенных элементов в строках и столбцах.
Итоговое значение нижней границы должно совпасть с длиной результирующего контура.
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <ctype.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define assert( bool ) 
int strempty(const char *p)
{
	if (!p)
		return (1);
	for (; *p; p++)
		if (!isspace(*p))
			return (0);
	return (1);
}
char *mystrtok(char **m, char *s, char c)
{
	char *p = s ? s : *m;
	if (!*p)
		return 0;
	*m = strchr(p, c);
	if (*m)
		*(*m)++ = 0;
	else
		*m = p + strlen(p);
	return p;
}
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

__global__ void global_queue_oneway_a(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 2*n; id += blockDim.x*gridDim.x) {
		islice[id] = 0;
		if (id < n){
			for (int i = 0; islice[id] < 2 && i < n; i++) {
				if (matrix[i*n + id] != LONG_MAX) {
					islice[id]++;
				}
			}
		}
		else {
			for (int j = 0; im[0] < 2 && j < n; j++) {
				if (matrix[(id - n)*n + j] != LONG_MAX) {
					islice[id]++;
				}
			}
		}
	}
}
__global__ void global_queue_oneway_b(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		for (int k = 0; k < 2 * n; k++){
			if (islice[k] == 1){
				if (k < n){
					int i; for (i = 0; i < n; i++){
						if (matrix[i*n + k] != LONG_MAX)
							break;
					}
					queue[--qsize[n]] = i*n + k;
				}
				else {
					int j; for (j = 0; j < n; j++){
						if (matrix[(k - n)*n + j] != LONG_MAX) 
							break;
					}
					queue[--qsize[n]] = (k - n)*n + j;
				}
			}
		}
	}
}
/*
Добавление запрещённых переходов.
	Шаг первый.
		Последнюю добавленную дугу помещаем в середину массива.
		Массив нарашиваем слева и справа.
	Шаг второй.
		Запрещаем все дуги ведущие из правой половины массива в левую половину массива.
*/
__global__ void global_add_forbidden_a(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		im[0] = im[1] = rank;
		islice[--im[0]] = islice[im[1]++] = n;
		while(1==1){
			int id1; for(id1 = rank; id1-->n ; ) if (to[id1]==from[islice[im[0]]]) break;
			if (id1>n) islice[--im[0]] = id1; else break;
		}
		while(1==1){
			int id2; for(id2 = rank; id2-->n ; ) if (from[id2]==to[islice[im[1]-1]]) break;
			if (id2>n) islice[im[1]++] = id2; else break;
		}
	}
}
__global__ void global_add_forbidden_b(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < (rank-im[0])*(im[1]-rank); id += blockDim.x*gridDim.x) {
		int id1 = islice[rank - (id%(rank-im[0])) - 1];
		int id2 = islice[rank + (id/(rank-im[0]))];
		int i; for (i = n; i-- > 0;) if (rows[i] == to[id2]) break; /* Номер строки */
		int j; for (j = n; j-- > 0;) if (cols[j] == from[id1]) break; /* Номер столбца */
		if (i != -1 && j != -1) matrix[i*n + j] = LONG_MAX;
	}
}
/*
Удаление строки im[0] и столбца im[1], соответствующих последней добавленной дуге (im[0],im[1])
*/
__global__ void global_matrix_trunc(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Удаляем строку и столбец параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < (n - 1)*(n - 1); id += blockDim.x*gridDim.x) {
		int i = id / (n - 1); /* Номер строки */
		int j = id % (n - 1); /* Номер столбца */
		if (i < im[0] && j < im[1]) matrix_1[id] = matrix[(i + 0)*n + j + 0];
		else if (i >= im[0] && j < im[1]) matrix_1[id] = matrix[(i + 1)*n + j + 0];
		else if (i < im[0] && j >= im[1]) matrix_1[id] = matrix[(i + 0)*n + j + 1];
		else if (i >= im[0] && j >= im[1]) matrix_1[id] = matrix[(i + 1)*n + j + 1];
	}
}
__global__ void global_queue_indexes_of_max(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Находим все индексы максимального коэффициента параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		for (int i = 0; i < (im[0] + 1); i++) {
			if (lm[1] == gamma[i]) queue[--qsize[n]] = i;
		}
	}
}
/*
Нахождение максимального индекса максимального элемента массива gamma
Возвращаемые значения:
	im[0] - индекс максимального элемента
	lm[1] - значение максимального элемента
*/
__global__ void global_gamma_max_index_of_max_a(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		islice[id] = id*n; 
		lslice[n + id] = gamma[islice[id]];
		for (int i = 1; i < n; i++) {
			if (lslice[n + id] <= gamma[id*n + i]) {
				islice[id] = id*n + i;
				lslice[n + id] = gamma[islice[id]];
			}
		}
	}
}
__global__ void global_gamma_max_index_of_max_b(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		im[0] = islice[0]; 
		lm[1] = lslice[n];
		for (int i = 1; i < n; i++) {
			if ((lm[1]  < lslice[n + i]) || ((lm[1] == lslice[n + i]) && (im[0] < islice[i]))) {
				im[0] = islice[i];
				lm[1] = lslice[n + i];
			}
		}
	}
}
/*
Для каждого нулевого элемента матрицы cij  рассчитаем коэффициент Гi,j, 
который равен сумме наименьшего элемента i строки (исключая элемент Сi,j=0) 
и наименьшего элемента j столбца.
Возвращаемые значения:
	gamma - массив рассчитанных коэффициентов

Массив gamma представляет собой расчёт минимальной цены въезда и выезда из города
*/
__global__ void global_calc_gamma(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Расчитываем коэффициенты параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		if (matrix[id] == 0) {
			int i = id / n; /* Номер строки */
			int j = id % n; /* Номер столбца */
			long x = matrix[i*n + ((j + 1) % n)]; /* Берём следующий элемент в качестве начального */
			long y = matrix[((i + 1) % n)*n + j]; /* Берём следующий элемент в качестве начального */
			for (int k = 2; k < n; k++){
				x = min(x, matrix[i*n + ((j + k) % n)]);
				y = min(y, matrix[((i + k) % n)*n + j]);
			}
			if ((x == LONG_MAX) && (y == LONG_MAX)) gamma[id] = LONG_MAX; /* Из города не въехать и не выехать */
			else if (x == LONG_MAX) gamma[id] = y; /* Из города не въехать */
			else if (y == LONG_MAX) gamma[id] = x; /* Из города не выехать */
			else gamma[id] = x + y;
		}
		else gamma[id] = LONG_MIN;
	}
}
__global__ void global_sub_by_row(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Находим минимальные значения в строках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		int i = id / n; /* Номер строки */
		if (matrix[id] != LONG_MAX) 
			matrix[id] -= lslice[i];
	}
}
__global__ void global_sub_by_col(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Находим минимальные значения в строках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		int j = id % n; /* Номер столбца */
		if (matrix[id] != LONG_MAX) 
			matrix[id] -= lslice[j];
	}
}
__global__ void global_min_by_col(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Находим минимальные значения в колонках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		lslice[id] = matrix[id];
		for (int i = 1; i < n; i++) {
			lslice[id] = min(lslice[id], matrix[i*n + id]);
		}
	}
}
__global__ void global_min_by_row(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Находим минимальные значения в строках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		lslice[id] = matrix[id*n];
		for (int j = 1; j < n; j++) {
			lslice[id] = min(lslice[id], matrix[id*n + j]);
		}
	}
}
__global__ void global_next_by_row(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		for (int i = 0; i < n; i++) {
			if (matrix[i*n + id] != LONG_MAX) {
				lslice[id] = max(lslice[id], lslice[i+n]);
				islice[id] = max(islice[id], islice[i+n]);
			}
		}
	}
}
__global__ void global_prev_by_col(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		for (int j = 0; j < n; j++) {
			if (matrix[id*n + j] != LONG_MAX){
				lslice[id] = max(lslice[id], lslice[j+n]);
				islice[id] = max(islice[id], islice[j+n]);
			}
		}
	}
}

__global__ void global_min_by_dim(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lm[0] = lslice[0];
		im[0] = islice[0];
		for (int i = 1; i < n; i++){
			lm[0] = min(lm[0], lslice[i]);
			im[0] = min(im[0], islice[i]);
		}
	}
}
__global__ void global_sum_lbound(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lm[0] = 0;
		for (int i = 1; i < n; i++) {
			lbound[i] = matrix[(n - 1)*i];
		}
		for (int i = 1; i <= rank; i++){
			lm[0] += lbound[i];
		}
	}
}
__global__ void global_add_lbound(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lbound[n] += matrix[queue[qsize[n]]];
	}
}
__global__ void global_sum_lbound_begin(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lbound[n] = 0;
	}
}
__global__ void global_sum_lbound_step(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		for (int i = 0; i < n; i++) lbound[n] += lslice[i];
	}
}
__global__ void global_slice_clear(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		islice[id] = 0;
		lslice[id] = 0;
	}
}
__global__ void global_sum_lbound_end(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
}
__global__ void global_check_infinity(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		im[0] = 0; for (int i = 0; im[0] == 0 && i < n; i++) if (lslice[i] == LONG_MAX) im[0] = 1;
	}
}
__global__ void global_initialize(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lbound[0] = 0;
		for (int i = 0; i < n; i++) rows[i] = i;
		for (int i = 0; i < n; i++) cols[i] = i;
		qsize[n + 1] = n*n*n;
		qsize[n] = qsize[n + 1];
	}
}
/*
	В случае неправильных параметров возвращённая лучшая цена имеет LONG_MAX значение
*/
__host__ void host_little(int gridSize, int blockSize, long *data, int *bestFrom, int *bestTo, long *bestPrice, int rank)
{
	hipError_t err;
	int n;         /* Ранг текущего массива */
	long **matrix;  /* Стек массивов элементов */
	int **rows;  /* Стек массивов элементов */
	int **cols;  /* Стек массивов элементов */
	long *gamma;    /* Массив коэффициентов */
	int *queue;    /* Стек очередей индексов элементов */
	int *qsize;    /* Размер очередей индексов элементов */
	long *lbound;   /* Стек вычисленных нижних границ */
	/* Стеки дуг (индексов) хранятся в порядке их удаления из матрицы */
	/* Индексы записаны в соответствии с текущим размером матрицы */
	/* и требуют пересчёта в исходный размер матрицы */
	int *from; /* Стек дуг (индексов) в порядке их удаления из матрицы */
	int *to;   /* Стек дуг (индексов) в порядке их удаления из матрицы */
	int *im;
	long *lm;
	int *islice;
	long *lslice;
	int ivalue[2];
	long lvalue[2];
	int *ibuffer;
	long *lbuffer;

	n = rank;

	ibuffer = (int*)malloc(n*n*sizeof(int));
	lbuffer = (long*)malloc(n*n*sizeof(long));
	matrix = (long**)malloc((n + 1)*sizeof(long*));
	rows = (int**)malloc((n + 1)*sizeof(int*));
	cols = (int**)malloc((n + 1)*sizeof(int*));
	for (int i = 1; i <= n; i++) err = hipMalloc((void**)&matrix[i], i*i*sizeof(long));
	for (int i = 1; i <= n; i++) err = hipMalloc((void**)&rows[i], i*sizeof(int));
	for (int i = 1; i <= n; i++) err = hipMalloc((void**)&cols[i], i*sizeof(int));

	err = hipMalloc((void**)&im, 2 * sizeof(int));
	err = hipMalloc((void**)&lm, 2 * sizeof(long));
	err = hipMalloc((void**)&islice, 2*n*sizeof(int));
	err = hipMalloc((void**)&lslice, 2*n*sizeof(long));
	err = hipMalloc((void**)&lbound ,(n + 1)*sizeof(long));
	err = hipMalloc((void**)&from, n*sizeof(int));
	err = hipMalloc((void**)&to, n*sizeof(int));
	err = hipMalloc((void**)&queue, n*n*n * sizeof(int));
	err = hipMalloc((void**)&qsize ,(n + 2)*sizeof(int));
	err = hipMalloc((void**)&gamma,n*n*sizeof(long));

	hipMemcpy(matrix[n], data, n*n*sizeof(long), hipMemcpyHostToDevice);

	global_initialize <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	*bestPrice = LONG_MAX;

	int blocks = (gridSize > 0)? gridSize : min(max(1, (int)pow((double)rank, 0.333333333333333)), 15);
	int threads = (blockSize > 0)? blockSize : min(max(1, (int)pow((double)rank, 0.333333333333333)), 15);

	ivalue[1] = 1;
	printf(" Check Graph by rows \n");
	/* Проверяем граф на связанность по строкам */
	global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(islice, &ivalue[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++)
	{
		hipMemcpy(&islice[n], islice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
		global_next_by_row <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	}
	hipMemcpy(ivalue, islice, sizeof(int), hipMemcpyDeviceToHost);
	if (ivalue[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}

	printf(" Check Graph by columns \n");
	/* Проверяем граф на связанность по столбцам */
	global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(islice, &ivalue[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++) {
		hipMemcpy(&islice[n], islice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
		global_prev_by_col <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	}
	hipMemcpy(ivalue, islice, sizeof(int), hipMemcpyDeviceToHost);
	if (ivalue[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}
	printf(" Check Graph by rows \n");
	/* Проверяем граф на связанность по строкам */
	global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(islice, &ivalue[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++)
	{
		hipMemcpy(&islice[n], islice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_next_by_row <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	}
	global_min_by_dim <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(ivalue, im, sizeof(int), hipMemcpyDeviceToHost);
	if (ivalue[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}

	printf(" Check Graph by columns \n");
	/* Проверяем граф на связанность по столбцам */
	global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(islice, &ivalue[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++) {
		hipMemcpy(&islice[n], islice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_prev_by_col <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	}
	global_min_by_dim <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(ivalue, im, sizeof(int), hipMemcpyDeviceToHost);
	if (ivalue[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}

	printf("Graph is ok\n");

	while (n > 0 && n <= rank) {

		
		printf("Matrix rank :\t%d\n", n);
		hipMemcpy(lbuffer, matrix[n], n*n*sizeof(long), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++){
			for (int j = 0; j < n; j++){
				printf("%ld%s", lbuffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
			}
		}

		int blocks0 = (gridSize > 0)? gridSize : min(max(1, (int)pow((double)(rank-n), 0.6666666666666)), 15);
		int threads0 = (blockSize > 0)? blockSize : min(max(1, (int)pow((double)(rank-n), 0.6666666666666)), 15);

		int blocks1 = (gridSize > 0)? gridSize : min(max(1, (int)pow((double)n, 0.333333333333333)), 15);
		int threads1 = (blockSize > 0)? blockSize : min(max(1, (int)pow((double)n, 0.333333333333333)), 15);

		int blocks2 = (gridSize > 0)? gridSize : min(max(1, (int)pow((double)n, 0.66666666666666)), 15);
		int threads2 = (blockSize > 0)? blockSize : min(max(1, (int)pow((double)n, 0.66666666666666)), 15);

		global_sum_lbound_begin <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

		if (n > 1)  {
			printf(" global_add_forbidden \n");
			/* Запрещаем обратные переходы */
			global_add_forbidden_a <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			global_add_forbidden_b <<< blocks0, threads0 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

			hipMemcpy(lbuffer, matrix[n], n*n*sizeof(long), hipMemcpyDeviceToHost);
			for (int i = 0; i < n; i++){
				for (int j = 0; j < n; j++){
					printf("%ld%s", lbuffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
				}
			}

			hipMemcpy(&qsize[n], &qsize[n + 1], sizeof(int), hipMemcpyDeviceToDevice);

			printf(" global_min_by_row \n");
			/* Находим минимальные значения в строках матрицы параллельно в процессах */
			global_min_by_row <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			global_check_infinity <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			hipMemcpy(ivalue, im, sizeof(int), hipMemcpyDeviceToHost);
			if (ivalue[0] == 0) {

				printf(" global_sub_by_row \n");
				/* Вычитаем минимальные значения из строк параллельно в процессах */
				global_sub_by_row <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

				hipMemcpy(lbuffer, matrix[n], n*n*sizeof(long), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++){
					for (int j = 0; j < n; j++){
						printf("%ld%s", lbuffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
					}
				}

				printf(" global_sum_lbound_step \n");
				global_sum_lbound_step <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			}

			printf(" global_min_by_col \n");
			/* Находим минимальные значения в столбцах матрицы параллельно в процессах */
			global_min_by_col <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			global_check_infinity <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			hipMemcpy(ivalue, im, sizeof(int), hipMemcpyDeviceToHost);
			if (ivalue[0] == 0) {

				printf(" global_sub_by_col \n");
				/* Вычитаем минимальные значения из столбцов параллельно в процессах */
				global_sub_by_col <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

				hipMemcpy(lbuffer, matrix[n], n*n*sizeof(long), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++){
					for (int j = 0; j < n; j++){
						printf("%ld%s", lbuffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
					}
				}

				printf(" global_sum_lbound_step \n");
				global_sum_lbound_step <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			}

			printf(" global_sum_lbound_end \n");
			global_sum_lbound_end <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

			hipMemcpy(lbuffer, &lbound[n], sizeof(long), hipMemcpyDeviceToHost);
			printf("%ld\n", lbuffer[0]);

			printf(" global_queue_oneway \n");
			/* Находим все индексы максимального коэффициента параллельно в процессах */
			global_queue_oneway_a <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			global_queue_oneway_b <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

			hipMemcpy(ivalue, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
			if (ivalue[1] > ivalue[0]) hipMemcpy(ibuffer, &queue[ivalue[0]], (ivalue[1] - ivalue[0])*sizeof(int), hipMemcpyDeviceToHost);
			for (int i = 0; i < (ivalue[1] - ivalue[0]); i++) printf("%d%s", ibuffer[i], (i == (ivalue[1] - ivalue[0]) - 1) ? "\n" : "\t");

			hipMemcpy(ivalue, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
			if (ivalue[0] == ivalue[1]) {
				printf(" global_calc_gamma \n");
				/* Расчитываем коэффициенты параллельно в процессах */
				global_calc_gamma <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

				hipMemcpy(lbuffer, gamma, n*n*sizeof(long), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++){
					for (int j = 0; j < n; j++){
						printf("%ld%s", lbuffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
					}
				}

				/* Находим максимальный индекс максимального коэффициента параллельно в процессах */
				global_gamma_max_index_of_max_a <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
				global_gamma_max_index_of_max_b <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

				hipMemcpy(lvalue, lm, 2 * sizeof(long), hipMemcpyDeviceToHost);
				if (lvalue[1] != LONG_MIN)
				{
					printf(" global_queue_indexes_of_max \n");
					/* Находим все индексы максимального коэффициента параллельно в процессах */
					global_queue_indexes_of_max <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

					hipMemcpy(ivalue, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
					if (ivalue[1] > ivalue[0]) hipMemcpy(ibuffer, &queue[ivalue[0]], (ivalue[1] - ivalue[0])*sizeof(int), hipMemcpyDeviceToHost);
					for (int i = 0; i < (ivalue[1] - ivalue[0]); i++) printf("%d%s", ibuffer[i], (i == (ivalue[1] - ivalue[0]) - 1) ? "\n" : "\t");

				}
			}
			else {
				ivalue[0] = ivalue[1] - 1;
				hipMemcpy(&qsize[n], ivalue, sizeof(int), hipMemcpyHostToDevice);
				printf(" global_add_lbound \n");
				global_add_lbound <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			}

			/* Теперь все индексы должны быть рекурсивно обработаны */
			/* Чтобы не делать рекурсивные обходы работаем только с объявленным стеком */
		}
		else {

			hipMemcpy(lvalue, matrix[n], sizeof(long), hipMemcpyDeviceToHost);
			if (lvalue[0] != LONG_MAX){
				hipMemcpy(from, rows[n], n*sizeof(int), hipMemcpyDeviceToDevice);
				hipMemcpy(to, cols[n], n*sizeof(int), hipMemcpyDeviceToDevice);

				printf(" global_sum_lbound \n");
				/* Суммируем Текущую Нижнюю Границу параллельно в процессах */
				global_sum_lbound <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);


				/* Сравниваем текущую стоимость с ранее найденой лучшей стоимостью */
				hipMemcpy(lvalue, lm, sizeof(long), hipMemcpyDeviceToHost);
				if (lvalue[0] < bestPrice[0]){
					bestPrice[0] = lvalue[0];
					hipMemcpy(bestFrom, from, rank * sizeof(int), hipMemcpyDeviceToHost);
					hipMemcpy(bestTo, to, rank * sizeof(int), hipMemcpyDeviceToHost);
				}
				printf("Current Price\t: %ld\n", bestPrice[0]);
			}
			n++;
		}

		/* Возврат из "рекурсивного" вызова */
		/* Чтобы не делать рекурсивные обходы работаем только с объявленным стеком */
		while ((n <= rank)) {
			hipMemcpy(ivalue, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
			if (ivalue[0] == ivalue[1]) {

				printf(" Return from Recursion \n");
				n++;
				continue;
			}
			break;
		}
		if (n > rank) break;

		/* Перебираем значения из очереди */
		hipMemcpy(ivalue, &qsize[n], sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&ivalue[1], &queue[ivalue[0]], sizeof(int), hipMemcpyDeviceToHost);
		ivalue[0]++;
		hipMemcpy(&qsize[n], ivalue, sizeof(int), hipMemcpyHostToDevice);

		int id = ivalue[1];
		ivalue[0] = id / n; /* Номер строки */
		ivalue[1] = id % n; /* Номер столбца */

		hipMemcpy(im, ivalue, 2 * sizeof(int), hipMemcpyHostToDevice);

		hipMemcpy(&from[n - 1], &rows[n][ivalue[0]], sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(&to[n - 1], &cols[n][ivalue[1]], sizeof(int), hipMemcpyDeviceToDevice);

		printf(" global_matrix_trunc \n");
		/* Удаляем строку и столбец */
		if (ivalue[0] > 0) hipMemcpy(rows[n - 1], rows[n], ivalue[0] * sizeof(int), hipMemcpyDeviceToDevice);
		if (ivalue[0] < (n - 1)) hipMemcpy(&rows[n - 1][ivalue[0]], &rows[n][ivalue[0] + 1], (n - ivalue[0] - 1) * sizeof(int), hipMemcpyDeviceToDevice);
		if (ivalue[1] > 0) hipMemcpy(cols[n - 1], cols[n], ivalue[1] * sizeof(int), hipMemcpyDeviceToDevice);
		if (ivalue[1] < (n - 1)) hipMemcpy(&cols[n - 1][ivalue[1]], &cols[n][ivalue[1] + 1], (n - ivalue[1] - 1) * sizeof(int), hipMemcpyDeviceToDevice);

		global_matrix_trunc <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

		n--;
	}
	n--;

the_end:
	/* Освобождаем ранее выделенные ресурсы */

	free(ibuffer);
	free(lbuffer);
	for (int i = 1; i <= n; i++) err = hipFree(matrix[i]);
	for (int i = 1; i <= n; i++) err = hipFree(rows[i]);
	for (int i = 1; i <= n; i++) err = hipFree(cols[i]);
	free(matrix);
	free(rows);
	free(cols);
	err = hipFree(gamma);
	err = hipFree(lbound);
	err = hipFree(queue);
	err = hipFree(qsize);
	err = hipFree(from);
	err = hipFree(to);
	err = hipFree(islice);
	err = hipFree(lslice);
	err = hipFree(im);
	err = hipFree(lm);

	err = err;
}

int main(int argc, char* argv[])
{
	int gridSize = 0;
	int blockSize = 0;

	printf("Title :\t%s\n", title); fflush(stdout);

	if (argc < 3) {
		printf("Usage :\t%s [-g <gridSize>] [-b <blockSize>] <inputfilename> <outputfilename>\n", argv[0]); fflush(stdout);
		printf("\tinputfilename - source matrix of path prices or empty\n"); fflush(stdout);
		printf("\toutputfilename - output best path point-to-point segments\n"); fflush(stdout);
		exit(-1);
	}

	int argId = 1;
	for(; argId < argc && argv[argId][0]=='-' ; argId++){
		switch(argv[argId][1]){
		case 'g':
			gridSize = atoi(argv[++argId]);
			break;
		case 'b':
			blockSize = atoi(argv[++argId]);
			break;
		}
	}

	char *inputFileName = argv[argId++];
	char *outputFileName = argv[argId++];

	char buffer[4096];
	char *tok;
	char *p;
	int n;         /* Ранг текущего массива */
	long *matrix;  /* Стек массивов элементов */
	int i, j;
	long bestPrice;
	int *bestFrom; /* Стек дуг (индексов) в порядке их удаления из матрицы */
	int *bestTo;   /* Стек дуг (индексов) в порядке их удаления из матрицы */

	printf("Input File Name :\t%s\n", inputFileName); fflush(stdout);
	printf("Output File Name :\t%s\n", outputFileName); fflush(stdout);

	FILE *fs = fopen(inputFileName, "r");
	if (fs == NULL) {
		fprintf(stderr, "File open error (%s)\n", inputFileName); fflush(stderr);
		exit(-1);
	}

	n = 0;

	/* Заполняем массив числами из файла */
	/* Операция выполняетя только на хост процессе */
	/* Операция выполняетя в два прохода по файлу */
	/* На первом проходе определяется ранг матрицы */
	/* На втором проходе считываются данные */
	for (i = 0; (tok = fgets(buffer, sizeof(buffer), fs)) != NULL; i++)
	{
		j = 0;
		for (tok = mystrtok(&p, tok, ';'); tok != NULL; tok = mystrtok(&p, NULL, ';'))
		{
			j++;
		}
		n = max(n, j);
	}
	n = max(n, i);

	matrix = (long *)malloc(n*n*sizeof(long));
	bestFrom = (int *)malloc((n + 1)*sizeof(int));
	bestTo = (int *)malloc((n + 1)*sizeof(int));

	fseek(fs, 0, SEEK_SET);

	for (i = 0; (tok = fgets(buffer, sizeof(buffer), fs)) != NULL; i++)
	{
		j = 0;
		for (tok = mystrtok(&p, tok, ';'); tok != NULL; tok = mystrtok(&p, NULL, ';'))
		{
			/* Пустые элементы - это запрещённые пути */
			matrix[n*i + j++] = strempty(tok) ? LONG_MAX : atol(tok);
		}
		for (; j < n; j++) matrix[n*i + j] = LONG_MAX;
	}
	for (j = 0; j < (n - i)*n; j++) matrix[n*i + j] = LONG_MAX;
	for (i = 0; i < n; i++) matrix[n*i + i] = LONG_MAX; /* Запрещаем петли */

	fclose(fs);

	printf("Matrix rank :\t%d\n", n);
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			printf("%ld%s", matrix[i*n + j], ((j == n - 1) ? "\n" : "\t"));
		}
	}
	fflush(stdout);

	// Find/set the device.
	int device_size = 0;
	hipGetDeviceCount(&device_size);
	for (i = 0; i < device_size; ++i)
	{
		hipDeviceProp_t hipDeviceProp_t;
		hipGetDeviceProperties(&hipDeviceProp_t, i);
		printf("Running on GPU %d (%s)\n", i, hipDeviceProp_t.name); 

		printf("Device has ECC support enabled %d\n",hipDeviceProp_t.ECCEnabled);
		printf("Number of asynchronous engines %d\n",hipDeviceProp_t.asyncEngineCount);
		printf("Device can map host memory with hipHostAlloc/hipHostGetDevicePointer %d\n",hipDeviceProp_t.canMapHostMemory);
		printf("Clock frequency in kilohertz %d\n",hipDeviceProp_t.clockRate);
		printf("Compute mode (See hipComputeMode) %d\n",hipDeviceProp_t.computeMode);
		printf("Device can possibly execute multiple kernels concurrently %d\n",hipDeviceProp_t.concurrentKernels);
		printf("Device can concurrently copy memory and execute a kernel. Deprecated. Use instead asyncEngineCount. %d\n",hipDeviceProp_t.deviceOverlap);
		printf("Device is integrated as opposed to discrete %d\n",hipDeviceProp_t.integrated);
		printf("Specified whether there is a run time limit on kernels %d\n",hipDeviceProp_t.kernelExecTimeoutEnabled);
		printf("Size of L2 cache in bytes %d\n",hipDeviceProp_t.l2CacheSize);
		printf("Major compute capability %d\n",hipDeviceProp_t.major);
		printf("Maximum size of each dimension of a grid %d\n",hipDeviceProp_t.maxGridSize[0]);
		printf("Maximum size of each dimension of a grid %d\n",hipDeviceProp_t.maxGridSize[1]);
		printf("Maximum size of each dimension of a grid %d\n",hipDeviceProp_t.maxGridSize[2]);
		printf("Maximum 1D surface size %d\n",hipDeviceProp_t.maxSurface1D);
		printf("Maximum 1D layered surface dimensions %d\n",hipDeviceProp_t.maxSurface1DLayered[0]);
		printf("Maximum 1D layered surface dimensions %d\n",hipDeviceProp_t.maxSurface1DLayered[1]);
		printf("Maximum 2D surface dimensions %d\n",hipDeviceProp_t.maxSurface2D[0]);
		printf("Maximum 2D surface dimensions %d\n",hipDeviceProp_t.maxSurface2D[1]);
		printf("Maximum 2D layered surface dimensions %d\n",hipDeviceProp_t.maxSurface2DLayered[0]);
		printf("Maximum 2D layered surface dimensions %d\n",hipDeviceProp_t.maxSurface2DLayered[1]);
		printf("Maximum 2D layered surface dimensions %d\n",hipDeviceProp_t.maxSurface2DLayered[2]);
		printf("Maximum 3D surface dimensions %d\n",hipDeviceProp_t.maxSurface3D[0]);
		printf("Maximum 3D surface dimensions %d\n",hipDeviceProp_t.maxSurface3D[1]);
		printf("Maximum 3D surface dimensions %d\n",hipDeviceProp_t.maxSurface3D[2]);
		printf("Maximum Cubemap surface dimensions %d\n",hipDeviceProp_t.maxSurfaceCubemap);
		printf("Maximum Cubemap layered surface dimensions %d\n",hipDeviceProp_t.maxSurfaceCubemapLayered[0]);
		printf("Maximum Cubemap layered surface dimensions %d\n",hipDeviceProp_t.maxSurfaceCubemapLayered[1]);
		printf("Maximum 1D texture size %d\n",hipDeviceProp_t.maxTexture1D);
		printf("Maximum 1D layered texture dimensions %d\n",hipDeviceProp_t.maxTexture1DLayered[0]);
		printf("Maximum 1D layered texture dimensions %d\n",hipDeviceProp_t.maxTexture1DLayered[1]);
		printf("Maximum size for 1D textures bound to linear memory %d\n",hipDeviceProp_t.maxTexture1DLinear);
		printf("Maximum 1D mipmapped texture size %d\n",hipDeviceProp_t.maxTexture1DMipmap);
		printf("Maximum 2D texture dimensions %d\n",hipDeviceProp_t.maxTexture2D[0]);
		printf("Maximum 2D texture dimensions %d\n",hipDeviceProp_t.maxTexture2D[1]);
		printf("Maximum 2D texture dimensions if texture gather operations have to be performed %d\n",hipDeviceProp_t.maxTexture2DGather[0]);
		printf("Maximum 2D texture dimensions if texture gather operations have to be performed %d\n",hipDeviceProp_t.maxTexture2DGather[1]);
		printf("Maximum 2D layered texture dimensions %d\n",hipDeviceProp_t.maxTexture2DLayered[0]);
		printf("Maximum 2D layered texture dimensions %d\n",hipDeviceProp_t.maxTexture2DLayered[1]);
		printf("Maximum 2D layered texture dimensions %d\n",hipDeviceProp_t.maxTexture2DLayered[2]);
		printf("Maximum dimensions (width, height, pitch) for 2D textures bound to pitched memory %d\n",hipDeviceProp_t.maxTexture2DLinear[0]);
		printf("Maximum dimensions (width, height, pitch) for 2D textures bound to pitched memory %d\n",hipDeviceProp_t.maxTexture2DLinear[1]);
		printf("Maximum dimensions (width, height, pitch) for 2D textures bound to pitched memory %d\n",hipDeviceProp_t.maxTexture2DLinear[2]);
		printf("Maximum 2D mipmapped texture dimensions %d\n",hipDeviceProp_t.maxTexture2DMipmap[0]);
		printf("Maximum 2D mipmapped texture dimensions %d\n",hipDeviceProp_t.maxTexture2DMipmap[1]);
		printf("Maximum 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3D[0]);
		printf("Maximum 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3D[1]);
		printf("Maximum 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3D[2]);
		printf("Maximum alternate 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3DAlt[0]);
		printf("Maximum alternate 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3DAlt[1]);
		printf("Maximum alternate 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3DAlt[2]);
		printf("Maximum Cubemap texture dimensions %d\n",hipDeviceProp_t.maxTextureCubemap);
		printf("Maximum Cubemap layered texture dimensions %d\n",hipDeviceProp_t.maxTextureCubemapLayered[0]);
		printf("Maximum Cubemap layered texture dimensions %d\n",hipDeviceProp_t.maxTextureCubemapLayered[1]);
		printf("Maximum size of each dimension of a block %d\n",hipDeviceProp_t.maxThreadsDim[0]);
		printf("Maximum size of each dimension of a block %d\n",hipDeviceProp_t.maxThreadsDim[1]);
		printf("Maximum size of each dimension of a block %d\n",hipDeviceProp_t.maxThreadsDim[2]);
		printf("Maximum number of threads per block %d\n",hipDeviceProp_t.maxThreadsPerBlock);
		printf("Maximum resident threads per multiprocessor %d\n",hipDeviceProp_t.maxThreadsPerMultiProcessor);
		printf("Maximum pitch in bytes allowed by memory copies %d\n",hipDeviceProp_t.memPitch);
		printf("Global memory bus width in bits %d\n",hipDeviceProp_t.memoryBusWidth);
		printf("Peak memory clock frequency in kilohertz %d\n",hipDeviceProp_t.memoryClockRate);
		printf("Minor compute capability %d\n",hipDeviceProp_t.minor);
		printf("Number of multiprocessors on device %d\n",hipDeviceProp_t.multiProcessorCount);
		printf("PCI bus ID of the device %d\n",hipDeviceProp_t.pciBusID);
		printf("PCI device ID of the device %d\n",hipDeviceProp_t.pciDeviceID);
		printf("PCI domain ID of the device %d\n",hipDeviceProp_t.pciDomainID);
		printf("32-bit registers available per block %d\n",hipDeviceProp_t.regsPerBlock);
		printf("Shared memory available per block in bytes %d\n",hipDeviceProp_t.sharedMemPerBlock);
		printf("Device supports stream priorities %d\n",hipDeviceProp_t.streamPrioritiesSupported);
		printf("Alignment requirements for surfaces %d\n",hipDeviceProp_t.surfaceAlignment);
		printf("1 if device is a Tesla device using TCC driver, 0 otherwise %d\n",hipDeviceProp_t.tccDriver);
		printf("Alignment requirement for textures %d\n",hipDeviceProp_t.textureAlignment);
		printf("Pitch alignment requirement for texture references bound to pitched memory %d\n",hipDeviceProp_t.texturePitchAlignment);
		printf("Constant memory available on device in bytes %d\n",hipDeviceProp_t.totalConstMem);
		printf("Global memory available on device in bytes %d\n",hipDeviceProp_t.totalGlobalMem);
		printf("Device shares a unified address space with the host %d\n",hipDeviceProp_t.unifiedAddressing);
		printf("Warp size in threads %d\n",hipDeviceProp_t.warpSize);

		fflush(stdout);
	}

	host_little(gridSize, blockSize, matrix, bestFrom, bestTo, &bestPrice, n);

	hipDeviceReset();

	/* Bыводим результаты */
	if (bestPrice != LONG_MAX){
		printf("Best Path\t: "); for (int i = 0; i < n; i++) printf("(%d,%d)%s", bestFrom[i], bestTo[i], ((i < (n - 1)) ? "," : "\n"));
		printf("Best Price\t: %ld\n", bestPrice);

		fs = fopen(outputFileName, "w");
		if (fs == NULL) {
			fprintf(stderr, "File open error (%s)\n", outputFileName); fflush(stderr);
			exit(-1);
		}
		for (int i = 0; i < n; i++) fprintf(fs, "%d;%d\n", bestFrom[i], bestTo[i]);
		fclose(fs);
	}


	free(matrix);
	free(bestFrom);
	free(bestTo);

	fflush(stdout);

	if (bestPrice == LONG_MAX) exit(-1);
	exit(0);
}