﻿char *title = "Little's algorithm";
char *description = "Алгоритм Литтла - метод решения задачи коммивояжера";
/*
Алгоритм Литтла применяют для поиска решения задачи коммивояжера в виде гамильтонова контура.
Данный алгоритм используется для поиска оптимального гамильтонова контура в графе, имеющем N вершин,
причем каждая вершина i связана с любой другой вершиной j двунаправленной дугой.
Каждой дуге приписан вес Сi,j, причем веса дуг строго положительны (Сi,j≥0).
Веса дуг образуют матрицу стоимости. Все элементы по диагонали матрицы приравнивают
к бесконечности (Сj,j=∞).

В случае, если пара вершин i и j не связана между собой (граф не полносвязный), то соответствующему элементу
матрицы стоимости приписываем вес, равный длине минимального пути между вершинами i и j.
Если в итоге дуга (i, j) войдет в результирующий контур, то ее необходимо заменить соответствующим ей путем.
Матрицу оптимальных путей между всеми вершинами графа можно получить применив алгоритм Данцига или Флойда.

Алгоритм Литтала является частным случаем применения метода "ветвей и границ" для конкретной задачи.
Общая идея тривиальна: нужно разделить огромное число перебираемых вариантов на классы и получить оценки
(снизу – в задаче минимизации, сверху – в задаче максимизации) для этих классов, чтобы иметь возможность
отбрасывать варианты не по одному, а целыми классами.
Трудность состоит в том, чтобы найти такое разделение на классы (ветви) и такие оценки (границы),
чтобы процедура была эффективной.

Алгоритм Литтла

В каждой строке матрицы стоимости найдем минимальный элемент и вычтем его из всех элементов строки.
Сделаем это и для столбцов.
Получим матрицу стоимости, каждая строка и каждый столбец которой содержат хотя бы один нулевой элемент.
Для каждого нулевого элемента матрицы cij  рассчитаем коэффициент Гi,j, который равен сумме наименьшего элемента i строки
(исключая элемент Сi,j=0) и наименьшего элемента j столбца.
Проверяем, что не существует однозначных путей - то есть с одним входом и выходом
Если такой путь есть, то выбираем его
иначе Из всех коэффициентов  Гi,j выберем такой, который является максимальным Гk,l=max{Гi,j}.
В гамильтонов контур вносится соответствующая дуга (k,l).
Удаляем k-тую строку и столбец l, поменяем на бесконечность значение элемента Сl,k (поскольку дуга (k,l) включена в контур,
то обратный путь из l в k недопустим).
Повторяем алгоритм шага 1, пока порядок матрицы не станет равным одному.
Получаем гамильтонов контур.
В ходе решения ведется постоянный подсчет текущего значения нижней границы.
Нижняя граница равна сумме всех вычтенных элементов в строках и столбцах.
Итоговое значение нижней границы должно совпасть с длиной результирующего контура.
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <ctype.h>
#include <limits.h>
#include <hip/hip_runtime.h>



#define assert( bool ) 
int strempty(const char *p)
{
	if (!p)
		return (1);
	for (; *p; p++)
		if (!isspace(*p))
			return (0);
	return (1);
}
char *mystrtok(char **m, char *s, char c)
{
	char *p = s ? s : *m;
	if (!*p)
		return 0;
	*m = strchr(p, c);
	if (*m)
		*(*m)++ = 0;
	else
		*m = p + strlen(p);
	return p;
}
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

__global__ void global_queue_oneway_a(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 2*n; id += blockDim.x*gridDim.x) {
		islice[id] = 0;
		if (id < n){
			for (int i = 0; islice[id] < 2 && i < n; i++) {
				if (matrix[i*n + id] != LONG_MAX) {
					islice[id]++;
				}
			}
		}
		else {
			for (int j = 0; im[0] < 2 && j < n; j++) {
				if (matrix[(id - n)*n + j] != LONG_MAX) {
					islice[id]++;
				}
			}
		}
	}
}
__global__ void global_queue_oneway_b(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		for (int k = 0; k < 2 * n; k++){
			if (islice[k] == 1){
				if (k < n){
					int i; for (i = 0; i < n; i++){
						if (matrix[i*n + k] != LONG_MAX)
							break;
					}
					queue[--qsize[n]] = i*n + k;
				}
				else {
					int j; for (j = 0; j < n; j++){
						if (matrix[(k - n)*n + j] != LONG_MAX) 
							break;
					}
					queue[--qsize[n]] = (k - n)*n + j;
				}
			}
		}
	}
}
__global__ void global_add_forbidden(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = n + blockDim.x*blockIdx.x + threadIdx.x; id < rank; id += blockDim.x*gridDim.x) {
		int i; for (i = n; i-- > 0;) if (rows[i] == to[id]) break; /* Номер строки */
		int j; for (j = n; j-- > 0;) if (cols[j] == from[id]) break; /* Номер столбца */
		if (i != -1 && j != -1) matrix[i*n + j] = LONG_MAX;
	}
}
__global__ void global_matrix_trunc(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Удаляем строку и столбец параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < (n - 1)*(n - 1); id += blockDim.x*gridDim.x) {
		int i = id / (n - 1); /* Номер строки */
		int j = id % (n - 1); /* Номер столбца */
		if (i < im[0] && j < im[1]) matrix_1[id] = matrix[(i + 0)*n + j + 0];
		else if (i >= im[0] && j < im[1]) matrix_1[id] = matrix[(i + 1)*n + j + 0];
		else if (i < im[0] && j >= im[1]) matrix_1[id] = matrix[(i + 0)*n + j + 1];
		else if (i >= im[0] && j >= im[1]) matrix_1[id] = matrix[(i + 1)*n + j + 1];
	}
}
__global__ void global_queue_indexes_of_max(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Находим все индексы максимального коэффициента параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		for (int i = 0; i < (im[0] + 1); i++) {
			if (lm[1] == gamma[i]) queue[--qsize[n]] = i;
		}
	}
}
__global__ void global_gamma_max_index_of_max_a(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		islice[id] = id*n; 
		lslice[n + id] = gamma[islice[id]];
		for (int i = 1; i < n; i++) {
			if (lslice[n + id] <= gamma[id*n + i]) {
				islice[id] = id*n + i;
				lslice[n + id] = gamma[islice[id]];
			}
		}
	}
}
__global__ void global_gamma_max_index_of_max_b(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		im[0] = islice[0]; 
		lm[1] = lslice[n];
		for (int i = 1; i < n; i++) {
			if ((lm[1]  < lslice[n + i]) || ((lm[1] == lslice[n + i]) && (im[0] < islice[i]))) {
				im[0] = islice[i];
				lm[1] = lslice[n + i];
			}
		}
	}
}
__global__ void global_calc_gamma(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Расчитываем коэффициенты параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		if (matrix[id] == 0) {
			int i = id / n; /* Номер строки */
			int j = id % n; /* Номер столбца */
			long x = matrix[i*n + ((j + 1) % n)]; /* Берём следующий элемент в качестве начального */
			long y = matrix[((i + 1) % n)*n + j]; /* Берём следующий элемент в качестве начального */
			for (int k = 2; k < n; k++){
				x = min(x, matrix[i*n + ((j + k) % n)]);
				y = min(y, matrix[((i + k) % n)*n + j]);
			}
			if ((x == LONG_MAX) && (y == LONG_MAX)) gamma[id] = LONG_MAX; /* Из города не въехать и не выехать */
			else if (x == LONG_MAX) gamma[id] = y; /* Из города не въехать */
			else if (y == LONG_MAX) gamma[id] = x; /* Из города не выехать */
			else gamma[id] = x + y;
		}
		else gamma[id] = LONG_MIN;
	}
}
__global__ void global_sub_by_row(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Находим минимальные значения в строках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		int i = id / n; /* Номер строки */
		if (matrix[id] != LONG_MAX) 
			matrix[id] -= lslice[i];
	}
}
__global__ void global_sub_by_col(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Находим минимальные значения в строках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		int j = id % n; /* Номер столбца */
		if (matrix[id] != LONG_MAX) 
			matrix[id] -= lslice[j];
	}
}
__global__ void global_min_by_col(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Находим минимальные значения в колонках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		lslice[id] = matrix[id];
		for (int i = 1; i < n; i++) {
			lslice[id] = min(lslice[id], matrix[i*n + id]);
		}
	}
}
__global__ void global_min_by_row(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	/* Находим минимальные значения в строках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		lslice[id] = matrix[id*n];
		for (int j = 1; j < n; j++) {
			lslice[id] = min(lslice[id], matrix[id*n + j]);
		}
	}
}
__global__ void global_next_by_row(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		for (int i = 0; i < n; i++) {
			if (matrix[i*n + id] != LONG_MAX) {
				lslice[id] = max(lslice[id], lslice[i+n]);
				islice[id] = max(islice[id], islice[i+n]);
			}
		}
	}
}
__global__ void global_prev_by_col(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		for (int j = 0; j < n; j++) {
			if (matrix[id*n + j] != LONG_MAX){
				lslice[id] = max(lslice[id], lslice[j+n]);
				islice[id] = max(islice[id], islice[j+n]);
			}
		}
	}
}

__global__ void global_min_by_dim(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lm[0] = lslice[0];
		im[0] = islice[0];
		for (int i = 1; i < n; i++){
			lm[0] = min(lm[0], lslice[i]);
			im[0] = min(im[0], islice[i]);
		}
	}
}
__global__ void global_sum_lbound(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lm[0] = 0;
		for (int i = 1; i < n; i++) {
			lbound[i] = matrix[(n - 1)*i];
		}
		for (int i = 1; i <= rank; i++){
			lm[0] += lbound[i];
		}
	}
}
__global__ void global_add_lbound(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lbound[n] += matrix[queue[qsize[n]]];
	}
}
__global__ void global_sum_lbound_begin(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lbound[n] = 0;
	}
}
__global__ void global_sum_lbound_step(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		for (int i = 0; i < n; i++) lbound[n] += lslice[i];
	}
}
__global__ void global_slice_clear(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		islice[id] = 0;
		lslice[id] = 0;
	}
}
__global__ void global_sum_lbound_end(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
}
__global__ void global_check_infinity(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		im[0] = 0; for (int i = 0; im[0] == 0 && i < n; i++) if (lslice[i] == LONG_MAX) im[0] = 1;
	}
}
__global__ void global_initialize(int *queue, int *qsize, long *lbound, long *gamma, int *islice, long *lslice, long *matrix_1, long *matrix, int *rows, int *cols, int *from, int *to, int *im, long *lm, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lbound[0] = 0;
		for (int i = 0; i < n; i++) rows[i] = i;
		for (int i = 0; i < n; i++) cols[i] = i;
		qsize[n + 1] = n*n*n;
		qsize[n] = qsize[n + 1];
	}
}
/*
	В случае неправильных параметров возвращённая лучшая цена имеет отрицвтельное значение
*/
__host__ void host_little(long *data, int *bestFrom, int *bestTo, long *bestPrice, int rank)
{
	hipError_t err;
	int n;         /* Ранг текущего массива */
	long **matrix;  /* Стек массивов элементов */
	int **rows;  /* Стек массивов элементов */
	int **cols;  /* Стек массивов элементов */
	long *gamma;    /* Массив коэффициентов */
	int *queue;    /* Стек очередей индексов элементов */
	int *qsize;    /* Размер очередей индексов элементов */
	long *lbound;   /* Стек вычисленных нижних границ */
	/* Стеки дуг (индексов) хранятся в порядке их удаления из матрицы */
	/* Индексы записаны в соответствии с текущим размером матрицы */
	/* и требуют пересчёта в исходный размер матрицы */
	int *from; /* Стек дуг (индексов) в порядке их удаления из матрицы */
	int *to;   /* Стек дуг (индексов) в порядке их удаления из матрицы */
	int *im;
	long *lm;
	int *islice;
	long *lslice;
	int ivalue[2];
	long lvalue[2];
	int *ibuffer;
	long *lbuffer;

	n = rank;

	ibuffer = (int*)malloc(n*n*sizeof(int));
	lbuffer = (long*)malloc(n*n*sizeof(long));
	matrix = (long**)malloc((n + 1)*sizeof(long*));
	rows = (int**)malloc((n + 1)*sizeof(int*));
	cols = (int**)malloc((n + 1)*sizeof(int*));
	for (int i = 1; i <= n; i++) err = hipMalloc((void**)&matrix[i], i*i*sizeof(long));
	for (int i = 1; i <= n; i++) err = hipMalloc((void**)&rows[i], i*sizeof(int));
	for (int i = 1; i <= n; i++) err = hipMalloc((void**)&cols[i], i*sizeof(int));

	err = hipMalloc((void**)&im, 2 * sizeof(int));
	err = hipMalloc((void**)&lm, 2 * sizeof(long));
	err = hipMalloc((void**)&islice, 2*n*sizeof(int));
	err = hipMalloc((void**)&lslice, 2*n*sizeof(long));
	err = hipMalloc((void**)&lbound ,(n + 1)*sizeof(long));
	err = hipMalloc((void**)&from, n*sizeof(int));
	err = hipMalloc((void**)&to, n*sizeof(int));
	err = hipMalloc((void**)&queue, n*n*n * sizeof(int));
	err = hipMalloc((void**)&qsize ,(n + 2)*sizeof(int));
	err = hipMalloc((void**)&gamma,n*n*sizeof(long));

	hipMemcpy(matrix[n], data, n*n*sizeof(int), hipMemcpyHostToDevice);

	global_initialize <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	*bestPrice = LONG_MAX;

	int blocks = min(max(1, (int)pow((double)rank, 0.333333333333333)), 15);
	int threads = min(max(1, (int)pow((double)rank, 0.333333333333333)), 15);

	ivalue[1] = 1;
	printf(" Check Graph by rows \n");
	/* Проверяем граф на связанность по строкам */
	global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(islice, &ivalue[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++)
	{
		hipMemcpy(&islice[n], islice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
		global_next_by_row <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	}
	hipMemcpy(ivalue, islice, sizeof(int), hipMemcpyDeviceToHost);
	if (ivalue[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}

	printf(" Check Graph by columns \n");
	/* Проверяем граф на связанность по столбцам */
	global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(islice, &ivalue[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++) {
		hipMemcpy(&islice[n], islice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
		global_prev_by_col <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	}
	hipMemcpy(ivalue, islice, sizeof(int), hipMemcpyDeviceToHost);
	if (ivalue[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}
	printf(" Check Graph by rows \n");
	/* Проверяем граф на связанность по строкам */
	global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(islice, &ivalue[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++)
	{
		hipMemcpy(&islice[n], islice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_next_by_row <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	}
	global_min_by_dim <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(ivalue, im, sizeof(int), hipMemcpyDeviceToHost);
	if (ivalue[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}

	printf(" Check Graph by columns \n");
	/* Проверяем граф на связанность по столбцам */
	global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(islice, &ivalue[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++) {
		hipMemcpy(&islice[n], islice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_prev_by_col <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	}
	global_min_by_dim <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
	hipMemcpy(ivalue, im, sizeof(int), hipMemcpyDeviceToHost);
	if (ivalue[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}

	printf("Graph is ok\n");

	while (n > 0 && n <= rank) {

		
		printf("Matrix rank :\t%d\n", n);
		hipMemcpy(lbuffer, matrix[n], n*n*sizeof(long), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++){
			for (int j = 0; j < n; j++){
				printf("%ld%s", lbuffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
			}
		}

		int blocks1 = min(max(1, (int)pow((double)n, 0.333333333333333)), 15);
		int threads1 = min(max(1, (int)pow((double)n, 0.333333333333333)), 15);

		int blocks2 = min(max(1, (int)pow((double)n, 0.66666666666666)), 15);
		int threads2 = min(max(1, (int)pow((double)n, 0.66666666666666)), 15);

		global_sum_lbound_begin <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

		printf(" global_add_forbidden \n");
		/* Запрещаем обратные переходы */
		global_add_forbidden <<< blocks, threads >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

		hipMemcpy(lbuffer, matrix[n], n*n*sizeof(long), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++){
			for (int j = 0; j < n; j++){
				printf("%ld%s", lbuffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
			}
		}

		if (n > 1)  {
			hipMemcpy(&qsize[n], &qsize[n + 1], sizeof(int), hipMemcpyDeviceToDevice);

			printf(" global_min_by_row \n");
			/* Находим минимальные значения в строках матрицы параллельно в процессах */
			global_min_by_row <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			global_check_infinity <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			hipMemcpy(ivalue, im, sizeof(int), hipMemcpyDeviceToHost);
			if (ivalue[0] == 0) {

				printf(" global_sub_by_row \n");
				/* Вычитаем минимальные значения из строк параллельно в процессах */
				global_sub_by_row <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

				hipMemcpy(lbuffer, matrix[n], n*n*sizeof(long), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++){
					for (int j = 0; j < n; j++){
						printf("%ld%s", lbuffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
					}
				}

				printf(" global_sum_lbound_step \n");
				global_sum_lbound_step <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			}

			printf(" global_min_by_col \n");
			/* Находим минимальные значения в столбцах матрицы параллельно в процессах */
			global_min_by_col <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			global_check_infinity <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			hipMemcpy(ivalue, im, sizeof(int), hipMemcpyDeviceToHost);
			if (ivalue[0] == 0) {

				printf(" global_sub_by_col \n");
				/* Вычитаем минимальные значения из столбцов параллельно в процессах */
				global_sub_by_col <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

				hipMemcpy(lbuffer, matrix[n], n*n*sizeof(long), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++){
					for (int j = 0; j < n; j++){
						printf("%ld%s", lbuffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
					}
				}

				printf(" global_sum_lbound_step \n");
				global_sum_lbound_step <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			}

			printf(" global_sum_lbound_end \n");
			global_sum_lbound_end <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

			hipMemcpy(lbuffer, &lbound[n], sizeof(long), hipMemcpyDeviceToHost);
			printf("%ld\n", lbuffer[0]);

			printf(" global_queue_oneway \n");
			/* Находим все индексы максимального коэффициента параллельно в процессах */
			global_queue_oneway_a <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			global_queue_oneway_b <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

			hipMemcpy(ivalue, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
			if (ivalue[1] > ivalue[0]) hipMemcpy(ibuffer, &queue[ivalue[0]], (ivalue[1] - ivalue[0])*sizeof(int), hipMemcpyDeviceToHost);
			for (int i = 0; i < (ivalue[1] - ivalue[0]); i++) printf("%d%s", ibuffer[i], (i == (ivalue[1] - ivalue[0]) - 1) ? "\n" : "\t");

			hipMemcpy(ivalue, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
			if (ivalue[0] == ivalue[1]) {
				printf(" global_calc_gamma \n");
				/* Расчитываем коэффициенты параллельно в процессах */
				global_calc_gamma <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

				hipMemcpy(lbuffer, gamma, n*n*sizeof(long), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++){
					for (int j = 0; j < n; j++){
						printf("%ld%s", lbuffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
					}
				}

				/* Находим максимальный индекс максимального коэффициента параллельно в процессах */
				global_gamma_max_index_of_max_a <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
				global_gamma_max_index_of_max_b <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

				hipMemcpy(lvalue, lm, 2 * sizeof(long), hipMemcpyDeviceToHost);
				if (lvalue[1] != LONG_MIN)
				{
					printf(" global_queue_indexes_of_max \n");
					/* Находим все индексы максимального коэффициента параллельно в процессах */
					global_queue_indexes_of_max <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

					hipMemcpy(ivalue, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
					if (ivalue[1] > ivalue[0]) hipMemcpy(ibuffer, &queue[ivalue[0]], (ivalue[1] - ivalue[0])*sizeof(int), hipMemcpyDeviceToHost);
					for (int i = 0; i < (ivalue[1] - ivalue[0]); i++) printf("%d%s", ibuffer[i], (i == (ivalue[1] - ivalue[0]) - 1) ? "\n" : "\t");

				}
			}
			else {
				ivalue[0] = ivalue[1] - 1;
				hipMemcpy(&qsize[n], ivalue, sizeof(int), hipMemcpyHostToDevice);
				printf(" global_add_lbound \n");
				global_add_lbound <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);
			}

			/* Теперь все индексы должны быть рекурсивно обработаны */
			/* Чтобы не делать рекурсивные обходы работаем только с объявленным стеком */
		}
		else {

			hipMemcpy(lvalue, matrix[n], sizeof(long), hipMemcpyDeviceToHost);
			if (lvalue[0] != LONG_MAX){
				hipMemcpy(from, rows[n], n*sizeof(int), hipMemcpyDeviceToDevice);
				hipMemcpy(to, cols[n], n*sizeof(int), hipMemcpyDeviceToDevice);

				printf(" global_sum_lbound \n");
				/* Суммируем Текущую Нижнюю Границу параллельно в процессах */
				global_sum_lbound <<< 1, 1 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);


				/* Сравниваем текущую стоимость с ранее найденой лучшей стоимостью */
				hipMemcpy(lvalue, lm, sizeof(long), hipMemcpyDeviceToHost);
				if (lvalue[0] < bestPrice[0]){
					bestPrice[0] = lvalue[0];
					hipMemcpy(bestFrom, from, rank * sizeof(int), hipMemcpyDeviceToHost);
					hipMemcpy(bestTo, to, rank * sizeof(int), hipMemcpyDeviceToHost);
				}
				printf("Current Price\t: %ld\n", bestPrice[0]);
			}
			n++;
		}

		/* Возврат из "рекурсивного" вызова */
		/* Чтобы не делать рекурсивные обходы работаем только с объявленным стеком */
		while ((n <= rank)) {
			hipMemcpy(ivalue, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
			if (ivalue[0] == ivalue[1]) {

				printf(" Return from Recursion \n");
				n++;
				continue;
			}
			break;
		}
		if (n > rank) break;

		/* Перебираем значения из очереди */
		hipMemcpy(ivalue, &qsize[n], sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&ivalue[1], &queue[ivalue[0]], sizeof(int), hipMemcpyDeviceToHost);
		ivalue[0]++;
		hipMemcpy(&qsize[n], ivalue, sizeof(int), hipMemcpyHostToDevice);

		int id = ivalue[1];
		ivalue[0] = id / n; /* Номер строки */
		ivalue[1] = id % n; /* Номер столбца */

		hipMemcpy(im, ivalue, 2 * sizeof(int), hipMemcpyHostToDevice);

		hipMemcpy(&from[n - 1], &rows[n][ivalue[0]], sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(&to[n - 1], &cols[n][ivalue[1]], sizeof(int), hipMemcpyDeviceToDevice);

		printf(" global_matrix_trunc \n");
		/* Удаляем строку и столбец */
		if (ivalue[0] > 0) hipMemcpy(rows[n - 1], rows[n], ivalue[0] * sizeof(int), hipMemcpyDeviceToDevice);
		if (ivalue[0] < (n - 1)) hipMemcpy(&rows[n - 1][ivalue[0]], &rows[n][ivalue[0] + 1], (n - ivalue[0] - 1) * sizeof(int), hipMemcpyDeviceToDevice);
		if (ivalue[1] > 0) hipMemcpy(cols[n - 1], cols[n], ivalue[1] * sizeof(int), hipMemcpyDeviceToDevice);
		if (ivalue[1] < (n - 1)) hipMemcpy(&cols[n - 1][ivalue[1]], &cols[n][ivalue[1] + 1], (n - ivalue[1] - 1) * sizeof(int), hipMemcpyDeviceToDevice);

		global_matrix_trunc <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, islice, lslice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, im, lm, n, rank);

		n--;
	}
	n--;

the_end:
	/* Освобождаем ранее выделенные ресурсы */

	free(ibuffer);
	free(lbuffer);
	for (int i = 1; i <= n; i++) hipFree(matrix[i]);
	for (int i = 1; i <= n; i++) hipFree(rows[i]);
	for (int i = 1; i <= n; i++) hipFree(cols[i]);
	free(matrix);
	free(rows);
	free(cols);
	hipFree(gamma);
	hipFree(lbound);
	hipFree(queue);
	hipFree(qsize);
	hipFree(from);
	hipFree(to);
	hipFree(islice);
	hipFree(lslice);
	hipFree(im);
	hipFree(lm);

	err = err;
}

int main(int argc, char* argv[])
{
	printf("Title :\t%s\n", title); fflush(stdout);

	if (argc < 3) {
		printf("Usage :\t%s <inputfilename> <outputfilename>\n", argv[0]); fflush(stdout);
		exit(-1);
	}

	char *inputFileName = argv[1];
	char *outputFileName = argv[2];

	char buffer[4096];
	char *tok;
	char *p;
	int n;         /* Ранг текущего массива */
	long *matrix;  /* Стек массивов элементов */
	int i, j;
	long bestPrice;
	int *bestFrom; /* Стек дуг (индексов) в порядке их удаления из матрицы */
	int *bestTo;   /* Стек дуг (индексов) в порядке их удаления из матрицы */

	printf("Input File Name :\t%s\n", inputFileName); fflush(stdout);
	printf("Output File Name :\t%s\n", outputFileName); fflush(stdout);

	FILE *fs = fopen(inputFileName, "r");
	if (fs == NULL) {
		fprintf(stderr, "File open error (%s)\n", inputFileName); fflush(stderr);
		exit(-1);
	}

	n = 0;

	/* Заполняем массив числами из файла */
	/* Операция выполняетя только на хост процессе */
	/* Операция выполняетя в два прохода по файлу */
	/* На первом проходе определяется ранг матрицы */
	/* На втором проходе считываются данные */
	for (i = 0; (tok = fgets(buffer, sizeof(buffer), fs)) != NULL; i++)
	{
		j = 0;
		for (tok = mystrtok(&p, tok, ';'); tok != NULL; tok = mystrtok(&p, NULL, ';'))
		{
			j++;
		}
		n = max(n, j);
	}
	n = max(n, i);

	matrix = (long *)malloc(n*n*sizeof(long));
	bestFrom = (int *)malloc((n + 1)*sizeof(int));
	bestTo = (int *)malloc((n + 1)*sizeof(int));

	fseek(fs, 0, SEEK_SET);

	for (i = 0; (tok = fgets(buffer, sizeof(buffer), fs)) != NULL; i++)
	{
		j = 0;
		for (tok = mystrtok(&p, tok, ';'); tok != NULL; tok = mystrtok(&p, NULL, ';'))
		{
			/* Пустые элементы - это запрещённые пути */
			matrix[n*i + j++] = strempty(tok) ? LONG_MAX : atol(tok);
		}
		for (; j < n; j++) matrix[n*i + j] = LONG_MAX;
	}
	for (j = 0; j < (n - i)*n; j++) matrix[n*i + j] = LONG_MAX;
	for (i = 0; i < n; i++) matrix[n*i + i] = LONG_MAX; /* Запрещаем петли */

	fclose(fs);

	printf("Matrix rank :\t%d\n", n);
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			printf("%ld%s", matrix[i*n + j], ((j == n - 1) ? "\n" : "\t"));
		}
	}
	fflush(stdout);

	// Find/set the device.
	int device_size = 0;
	hipGetDeviceCount(&device_size);
	for (i = 0; i < device_size; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		printf("Running on GPU %d (%s)\n", i, properties.name); fflush(stdout);
	}

	host_little(matrix, bestFrom, bestTo, &bestPrice, n);

	hipDeviceReset();

	/* Bыводим результаты */
	if (bestPrice != LONG_MAX){
		printf("Best Path\t: "); for (int i = 0; i < n; i++) printf("(%d,%d)%s", bestFrom[i], bestTo[i], ((i < (n - 1)) ? "," : "\n"));
		printf("Best Price\t: %ld\n", bestPrice);

		fs = fopen(outputFileName, "w");
		if (fs == NULL) {
			fprintf(stderr, "File open error (%s)\n", outputFileName); fflush(stderr);
			exit(-1);
		}
		for (int i = 0; i < n; i++) fprintf(fs, "%d;%d\n", bestFrom[i], bestTo[i]);
		fclose(fs);
	}


	free(matrix);
	free(bestFrom);
	free(bestTo);

	fflush(stdout);

	if (bestPrice == LONG_MAX) exit(-1);
	exit(0);
}