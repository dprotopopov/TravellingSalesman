#include "hip/hip_runtime.h"
﻿char *title = "Little's algorithm";
char *description = "Алгоритм Литтла - метод решения задачи коммивояжера";
/*
Алгоритм Литтла применяют для поиска решения задачи коммивояжера в виде гамильтонова контура.
Данный алгоритм используется для поиска оптимального гамильтонова контура в графе, имеющем N вершин,
причем каждая вершина i связана с любой другой вершиной j двунаправленной дугой.
Каждой дуге приписан вес Сi,j, причем веса дуг строго положительны (Сi,j≥0).
Веса дуг образуют матрицу стоимости. Все элементы по диагонали матрицы приравнивают
к бесконечности (Сj,j=∞).

В случае, если пара вершин i и j не связана между собой (граф не полносвязный), то соответствующему элементу
матрицы стоимости приписываем вес, равный длине минимального пути между вершинами i и j.
Если в итоге дуга (i, j) войдет в результирующий контур, то ее необходимо заменить соответствующим ей путем.
Матрицу оптимальных путей между всеми вершинами графа можно получить применив алгоритм Данцига или Флойда.

Алгоритм Литтала является частным случаем применения метода "ветвей и границ" для конкретной задачи.
Общая идея тривиальна: нужно разделить огромное число перебираемых вариантов на классы и получить оценки
(снизу – в задаче минимизации, сверху – в задаче максимизации) для этих классов, чтобы иметь возможность
отбрасывать варианты не по одному, а целыми классами.
Трудность состоит в том, чтобы найти такое разделение на классы (ветви) и такие оценки (границы),
чтобы процедура была эффективной.

Алгоритм Литтла

В каждой строке матрицы стоимости найдем минимальный элемент и вычтем его из всех элементов строки.
Сделаем это и для столбцов.
Получим матрицу стоимости, каждая строка и каждый столбец которой содержат хотя бы один нулевой элемент.
Для каждого нулевого элемента матрицы cij  рассчитаем коэффициент Гi,j, который равен сумме наименьшего элемента i строки
(исключая элемент Сi,j=0) и наименьшего элемента j столбца.
Проверяем, что не существует однозначных путей - то есть с одним входом и выходом
Если такой путь есть, то выбираем его
иначе Из всех коэффициентов  Гi,j выберем такой, который является максимальным Гk,l=max{Гi,j}.
В гамильтонов контур вносится соответствующая дуга (k,l).
Удаляем k-тую строку и столбец l, поменяем на бесконечность значение элемента Сl,k (поскольку дуга (k,l) включена в контур,
то обратный путь из l в k недопустим).
Повторяем алгоритм шага 1, пока порядок матрицы не станет равным одному.
Получаем гамильтонов контур.
В ходе решения ведется постоянный подсчет текущего значения нижней границы.
Нижняя граница равна сумме всех вычтенных элементов в строках и столбцах.
Итоговое значение нижней границы должно совпасть с длиной результирующего контура.
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <ctype.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define assert( bool ) 
int strempty(const char *p)
{
	if (!p)
		return (1);
	for (; *p; p++)
		if (!isspace(*p))
			return (0);
	return (1);
}
char *mystrtok(char **m, char *s, char c)
{
	char *p = s ? s : *m;
	if (!*p)
		return 0;
	*m = strchr(p, c);
	if (*m)
		*(*m)++ = 0;
	else
		*m = p + strlen(p);
	return p;
}
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

__global__ void global_queue_oneway_a(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 2*n; id += blockDim.x*gridDim.x) {
		slice[id] = 0;
		if (id < n){
			for (int i = 0; slice[id] < 2 && i < n; i++) {
				if (matrix[i*n + id] != INT_MAX) {
					slice[id]++;
				}
			}
		}
		else {
			for (int j = 0; m[0] < 2 && j < n; j++) {
				if (matrix[(id - n)*n + j] != INT_MAX) {
					slice[id]++;
				}
			}
		}
	}
}
__global__ void global_queue_oneway_b(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		for (int k = 0; k < 2 * n; k++){
			if (slice[k] == 1){
				if (k < n){
					int i; for (i = 0; i < n; i++){
						if (matrix[i*n + k] != INT_MAX)
							break;
					}
					queue[--qsize[n]] = i*n + k;
				}
				else {
					int j; for (j = 0; j < n; j++){
						if (matrix[(k - n)*n + j] != INT_MAX) 
							break;
					}
					queue[--qsize[n]] = (k - n)*n + j;
				}
			}
		}
	}
}
__global__ void global_add_forbidden(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = n + blockDim.x*blockIdx.x + threadIdx.x; id < rank; id += blockDim.x*gridDim.x) {
		int i; for (i = n; i-- > 0;) if (rows[i] == to[id]) break; /* Номер строки */
		int j; for (j = n; j-- > 0;) if (cols[j] == from[id]) break; /* Номер столбца */
		if (i != -1 && j != -1) matrix[i*n + j] = INT_MAX;
	}
}
__global__ void global_matrix_trunc(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	/* Удаляем строку и столбец параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < (n - 1)*(n - 1); id += blockDim.x*gridDim.x) {
		int i = id / (n - 1); /* Номер строки */
		int j = id % (n - 1); /* Номер столбца */
		if (i < m[0] && j < m[1]) matrix_1[id] = matrix[(i + 0)*n + j + 0];
		else if (i >= m[0] && j < m[1]) matrix_1[id] = matrix[(i + 1)*n + j + 0];
		else if (i < m[0] && j >= m[1]) matrix_1[id] = matrix[(i + 0)*n + j + 1];
		else if (i >= m[0] && j >= m[1]) matrix_1[id] = matrix[(i + 1)*n + j + 1];
	}
}
__global__ void global_queue_indexes_of_max(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	/* Находим все индексы максимального коэффициента параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		for (int i = 0; i < (m[0] + 1); i++) {
			if (m[1] == gamma[i]) queue[--qsize[n]] = i;
		}
	}
}
__global__ void global_gamma_max_index_of_max_a(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		slice[id] = id*n; slice[n + id] = gamma[slice[id]];
		for (int i = 1; i < n; i++) {
			if (slice[n + id] <= gamma[id*n + i]) {
				slice[id] = id*n + i;
				slice[n + id] = gamma[slice[id]];
			}
		}
	}
}
__global__ void global_gamma_max_index_of_max_b(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		m[0] = slice[0]; m[1] = slice[n];
		for (int i = 1; i < n; i++) {
			if ((m[1]  < slice[n + i]) || ((m[1] == slice[n + i]) && (m[0] < slice[i]))) {
				m[0] = slice[i];
				m[1] = slice[n + i];
			}
		}
	}
}
__global__ void global_calc_gamma(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	/* Расчитываем коэффициенты параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		if (matrix[id] == 0) {
			int i = id / n; /* Номер строки */
			int j = id % n; /* Номер столбца */
			int x = matrix[i*n + ((j + 1) % n)]; /* Берём следующий элемент в качестве начального */
			int y = matrix[((i + 1) % n)*n + j]; /* Берём следующий элемент в качестве начального */
			for (int k = 2; k < n; k++){
				x = min(x, matrix[i*n + ((j + k) % n)]);
				y = min(y, matrix[((i + k) % n)*n + j]);
			}
			if ((x == INT_MAX) && (y == INT_MAX)) gamma[id] = INT_MAX; /* Из города не въехать и не выехать */
			else if (x == INT_MAX) gamma[id] = y; /* Из города не въехать */
			else if (y == INT_MAX) gamma[id] = x; /* Из города не выехать */
			else gamma[id] = x + y;
		}
		else gamma[id] = INT_MIN;
	}
}
__global__ void global_sub_by_row(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	/* Находим минимальные значения в строках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		int i = id / n; /* Номер строки */
		if (matrix[id] != INT_MAX) 
			matrix[id] -= slice[i];
	}
}
__global__ void global_sub_by_col(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	/* Находим минимальные значения в строках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		int j = id % n; /* Номер столбца */
		if (matrix[id] != INT_MAX) 
			matrix[id] -= slice[j];
	}
}
__global__ void global_min_by_col(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	/* Находим минимальные значения в колонках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		slice[id] = matrix[id];
		for (int i = 1; i < n; i++) {
			slice[id] = min(slice[id], matrix[i*n + id]);
		}
	}
}
__global__ void global_min_by_row(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	/* Находим минимальные значения в строках матрицы параллельно в процессах */
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		slice[id] = matrix[id*n];
		for (int j = 1; j < n; j++) {
			slice[id] = min(slice[id], matrix[id*n + j]);
		}
	}
}
__global__ void global_next_by_row(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		for (int i = 0; i < n; i++) {
			if (matrix[i*n + id] != INT_MAX) {
				slice[id] = max(slice[id], slice[i+n]);
			}
		}
	}
}
__global__ void global_prev_by_col(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		for (int j = 0; j < n; j++) {
			if (matrix[id*n + j] != INT_MAX){
				slice[id] = max(slice[id], slice[j+n]);
			}
		}
	}
}

__global__ void global_min_by_dim(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		m[0] = slice[0];
		for (int i = 1; i < n; i++){
			m[0] = min(m[0], slice[i]);
		}
	}
}
__global__ void global_sum_lbound(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		m[0] = 0;
		for (int i = 1; i < n; i++) {
			lbound[i] = matrix[(n - 1)*i];
		}
		for (int i = 1; i <= rank; i++){
			m[0] += lbound[i];
		}
	}
}
__global__ void global_add_lbound(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lbound[n] += matrix[queue[n]];
	}
}
__global__ void global_sum_lbound_begin(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lbound[n] = 0;
	}
}
__global__ void global_sum_lbound_step(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		for (int i = 0; i < n; i++) lbound[n] += slice[i];
	}
}
__global__ void global_slice_clear(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n; id += blockDim.x*gridDim.x) {
		slice[id] = 0;
	}
}
__global__ void global_sum_lbound_end(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
}
__global__ void global_check_infinity(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		m[0] = 0; for (int i = 0; m[0] == 0 && i < n; i++) if (slice[i] == INT_MAX) m[0] = 1;
	}
}
__global__ void global_initialize(int *queue, int *qsize, int *lbound, int *gamma, int *slice, int *matrix_1, int *matrix, int *rows, int *cols, int *from, int *to, int *m, int n, int rank){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < 1; id += blockDim.x*gridDim.x) {
		lbound[0] = 0;
		for (int i = 0; i < n; i++) rows[i] = i;
		for (int i = 0; i < n; i++) cols[i] = i;
		qsize[n + 1] = n*n*n;
		qsize[n] = qsize[n + 1];
	}
}
/*
	В случае неправильных параметров возвращённая лучшая цена имеет отрицвтельное значение
*/
__host__ void host_little(int *data, int *bestFrom, int *bestTo, int *bestPrice, int rank)
{
	hipError_t err;
	int n;         /* Ранг текущего массива */
	int **matrix;  /* Стек массивов элементов */
	int **rows;  /* Стек массивов элементов */
	int **cols;  /* Стек массивов элементов */
	int *gamma;    /* Массив коэффициентов */
	int *queue;    /* Стек очередей индексов элементов */
	int *qsize;    /* Размер очередей индексов элементов */
	int *lbound;   /* Стек вычисленных нижних границ */
	/* Стеки дуг (индексов) хранятся в порядке их удаления из матрицы */
	/* Индексы записаны в соответствии с текущим размером матрицы */
	/* и требуют пересчёта в исходный размер матрицы */
	int *from; /* Стек дуг (индексов) в порядке их удаления из матрицы */
	int *to;   /* Стек дуг (индексов) в порядке их удаления из матрицы */
	int *m;
	int *slice;
	int value[2];
	int *buffer;

	n = rank;

	buffer = (int*)malloc(n*n*sizeof(int));
	matrix = (int**)malloc((n + 1)*sizeof(int*));
	rows = (int**)malloc((n + 1)*sizeof(int*));
	cols = (int**)malloc((n + 1)*sizeof(int*));
	for (int i = 1; i <= n; i++) err = hipMalloc((void**)&matrix[i], i*i*sizeof(int));
	for (int i = 1; i <= n; i++) err = hipMalloc((void**)&rows[i], i*sizeof(int));
	for (int i = 1; i <= n; i++) err = hipMalloc((void**)&cols[i], i*sizeof(int));

	err = hipMalloc((void**)&m, 2 * sizeof(int));
	err = hipMalloc((void**)&slice, 2*n*sizeof(int));
	err = hipMalloc((void**)&lbound ,(n + 1)*sizeof(int));
	err = hipMalloc((void**)&from, n*sizeof(int));
	err = hipMalloc((void**)&to, n*sizeof(int));
	err = hipMalloc((void**)&queue, n*n*n * sizeof(int));
	err = hipMalloc((void**)&qsize ,(n + 2)*sizeof(int));
	err = hipMalloc((void**)&gamma,n*n*sizeof(int));

	hipMemcpy(matrix[n], data, n*n*sizeof(int), hipMemcpyHostToDevice);

	global_initialize <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	*bestPrice = INT_MAX;

	int blocks = min(max(1, (int)pow((double)rank, 0.333333333333333)), 15);
	int threads = min(max(1, (int)pow((double)rank, 0.333333333333333)), 15);

	value[1] = 1;
	printf(" Check Graph by rows \n");
	/* Проверяем граф на связанность по строкам */
	global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	hipMemcpy(slice, &value[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++)
	{
		hipMemcpy(&slice[n], slice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
		global_next_by_row <<< blocks, threads >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	}
	hipMemcpy(value, slice, sizeof(int), hipMemcpyDeviceToHost);
	if (value[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}

	printf(" Check Graph by columns \n");
	/* Проверяем граф на связанность по столбцам */
	global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	hipMemcpy(slice, &value[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++) {
		hipMemcpy(&slice[n], slice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_slice_clear <<< blocks, threads >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
		global_prev_by_col <<< blocks, threads >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	}
	hipMemcpy(value, slice, sizeof(int), hipMemcpyDeviceToHost);
	if (value[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}
	printf(" Check Graph by rows \n");
	/* Проверяем граф на связанность по строкам */
	global_slice_clear << < blocks, threads >> >(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	hipMemcpy(slice, &value[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++)
	{
		hipMemcpy(&slice[n], slice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_next_by_row <<< blocks, threads >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	}
	global_min_by_dim << < 1, 1 >> >(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	hipMemcpy(value, m, sizeof(int), hipMemcpyDeviceToHost);
	if (value[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}

	printf(" Check Graph by columns \n");
	/* Проверяем граф на связанность по столбцам */
	global_slice_clear << < blocks, threads >> >(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	hipMemcpy(slice, &value[1], sizeof(int), hipMemcpyHostToDevice);
	for (int i = 1; i <= n; i++) {
		hipMemcpy(&slice[n], slice, n*sizeof(int), hipMemcpyDeviceToDevice);
		global_prev_by_col <<< blocks, threads >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	}
	global_min_by_dim << < 1, 1 >> >(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
	hipMemcpy(value, m, sizeof(int), hipMemcpyDeviceToHost);
	if (value[0] == 0) {
		fprintf(stderr, "Wrong Graph\n"); fflush(stderr);
		goto the_end;
	}

	printf("Graph is ok\n");

	while (n > 0 && n <= rank) {

		
		printf("Matrix rank :\t%d\n", n);
		hipMemcpy(buffer, matrix[n], n*n*sizeof(int), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++){
			for (int j = 0; j < n; j++){
				printf("%d%s", buffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
			}
		}

		int blocks1 = min(max(1, (int)pow((double)n, 0.333333333333333)), 15);
		int threads1 = min(max(1, (int)pow((double)n, 0.333333333333333)), 15);

		int blocks2 = min(max(1, (int)pow((double)n, 0.66666666666666)), 15);
		int threads2 = min(max(1, (int)pow((double)n, 0.66666666666666)), 15);

		global_sum_lbound_begin <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);

		printf(" global_add_forbidden \n");
		/* Запрещаем обратные переходы */
		global_add_forbidden <<< blocks, threads >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);

		hipMemcpy(buffer, matrix[n], n*n*sizeof(int), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++){
			for (int j = 0; j < n; j++){
				printf("%d%s", buffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
			}
		}

		if (n > 1)  {
			hipMemcpy(&qsize[n], &qsize[n + 1], sizeof(int), hipMemcpyDeviceToDevice);

			printf(" global_min_by_row \n");
			/* Находим минимальные значения в строках матрицы параллельно в процессах */
			global_min_by_row <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
			global_check_infinity <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
			hipMemcpy(value, m, sizeof(int), hipMemcpyDeviceToHost);
			if (value[0] == 0) {

				printf(" global_sub_by_row \n");
				/* Вычитаем минимальные значения из строк параллельно в процессах */
				global_sub_by_row <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);

				hipMemcpy(buffer, matrix[n], n*n*sizeof(int), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++){
					for (int j = 0; j < n; j++){
						printf("%d%s", buffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
					}
				}

				printf(" global_sum_lbound_step \n");
				global_sum_lbound_step <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
			}

			printf(" global_min_by_col \n");
			/* Находим минимальные значения в столбцах матрицы параллельно в процессах */
			global_min_by_col <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
			global_check_infinity <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
			hipMemcpy(value, m, sizeof(int), hipMemcpyDeviceToHost);
			if (value[0] == 0) {

				printf(" global_sub_by_col \n");
				/* Вычитаем минимальные значения из столбцов параллельно в процессах */
				global_sub_by_col <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);

				hipMemcpy(buffer, matrix[n], n*n*sizeof(int), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++){
					for (int j = 0; j < n; j++){
						printf("%d%s", buffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
					}
				}

				printf(" global_sum_lbound_step \n");
				global_sum_lbound_step <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
			}

			printf(" global_sum_lbound_end \n");
			global_sum_lbound_end <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);

			hipMemcpy(buffer, &lbound[n], sizeof(int), hipMemcpyDeviceToHost);
			printf("%d\n", buffer[0]);

			printf(" global_queue_oneway \n");
			/* Находим все индексы максимального коэффициента параллельно в процессах */
			global_queue_oneway_a <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
			global_queue_oneway_b <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);

			hipMemcpy(value, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
			if (value[1] > value[0]) hipMemcpy(buffer, &queue[value[0]], (value[1] - value[0])*sizeof(int), hipMemcpyDeviceToHost);
			for (int i = 0; i < (value[1] - value[0]); i++) printf("%d%s", buffer[i], (i == (value[1] - value[0]) - 1) ? "\n" : "\t");

			hipMemcpy(value, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
			if (value[0] == value[1]) {
				printf(" global_calc_gamma \n");
				/* Расчитываем коэффициенты параллельно в процессах */
				global_calc_gamma <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);

				hipMemcpy(buffer, gamma, n*n*sizeof(int), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++){
					for (int j = 0; j < n; j++){
						printf("%d%s", buffer[i*n + j], ((j == n - 1) ? "\n" : "\t"));
					}
				}

				/* Находим максимальный индекс максимального коэффициента параллельно в процессах */
				global_gamma_max_index_of_max_a <<< blocks1, threads1 >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
				global_gamma_max_index_of_max_b <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);

				hipMemcpy(value, m, 2 * sizeof(int), hipMemcpyDeviceToHost);
				if (value[1] != INT_MIN)
				{
					printf(" global_queue_indexes_of_max \n");
					/* Находим все индексы максимального коэффициента параллельно в процессах */
					global_queue_indexes_of_max <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);

					hipMemcpy(value, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
					if (value[1] > value[0]) hipMemcpy(buffer, &queue[value[0]], (value[1] - value[0])*sizeof(int), hipMemcpyDeviceToHost);
					for (int i = 0; i < (value[1] - value[0]); i++) printf("%d%s", buffer[i], (i == (value[1] - value[0]) - 1) ? "\n" : "\t");

				}
			}
			else {
				value[0] = value[1] - 1;
				hipMemcpy(&qsize[n], value, sizeof(int), hipMemcpyHostToDevice);
				printf(" global_add_lbound \n");
				global_add_lbound <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);
			}

			/* Теперь все индексы должны быть рекурсивно обработаны */
			/* Чтобы не делать рекурсивные обходы работаем только с объявленным стеком */
		}
		else {

			hipMemcpy(value, matrix[n], sizeof(int), hipMemcpyDeviceToHost);
			if (value[0] != INT_MAX){
				hipMemcpy(from, rows[n], n*sizeof(int), hipMemcpyDeviceToDevice);
				hipMemcpy(to, cols[n], n*sizeof(int), hipMemcpyDeviceToDevice);

				printf(" global_sum_lbound \n");
				/* Суммируем Текущую Нижнюю Границу параллельно в процессах */
				global_sum_lbound <<< 1, 1 >>>(queue, qsize, lbound, gamma, slice, matrix[n - 1], matrix[n], rows[n], cols[n], from, to, m, n, rank);


				/* Сравниваем текущую стоимость с ранее найденой лучшей стоимостью */
				hipMemcpy(value, m, sizeof(int), hipMemcpyDeviceToHost);
				if (value[0] < bestPrice[0]){
					bestPrice[0] = value[0];
					hipMemcpy(bestFrom, from, rank * sizeof(int), hipMemcpyDeviceToHost);
					hipMemcpy(bestTo, to, rank * sizeof(int), hipMemcpyDeviceToHost);
				}
				printf("Current Price\t: %d\n", bestPrice[0]);
			}
			n++;
		}

		/* Возврат из "рекурсивного" вызова */
		/* Чтобы не делать рекурсивные обходы работаем только с объявленным стеком */
		while ((n <= rank)) {
			hipMemcpy(value, &qsize[n], 2 * sizeof(int), hipMemcpyDeviceToHost);
			if (value[0] == value[1]) {

				printf(" Return from Recursion \n");
				n++;
				continue;
			}
			break;
		}
		if (n > rank) break;

		/* Перебираем значения из очереди */
		hipMemcpy(value, &qsize[n], sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&value[1], &queue[value[0]], sizeof(int), hipMemcpyDeviceToHost);
		value[0]++;
		hipMemcpy(&qsize[n], value, sizeof(int), hipMemcpyHostToDevice);

		int id = value[1];
		value[0] = id / n; /* Номер строки */
		value[1] = id % n; /* Номер столбца */

		hipMemcpy(m, value, 2 * sizeof(int), hipMemcpyHostToDevice);

		hipMemcpy(&from[n - 1], &rows[n][value[0]], sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(&to[n - 1], &cols[n][value[1]], sizeof(int), hipMemcpyDeviceToDevice);

		printf(" global_matrix_trunc \n");
		/* Удаляем строку и столбец */
		if (value[0] > 0) hipMemcpy(rows[n - 1], rows[n], value[0] * sizeof(int), hipMemcpyDeviceToDevice);
		if (value[0] < (n - 1)) hipMemcpy(&rows[n - 1][value[0]], &rows[n][value[0] + 1], (n - value[0] - 1) * sizeof(int), hipMemcpyDeviceToDevice);
		if (value[1] > 0) hipMemcpy(cols[n - 1], cols[n], value[1] * sizeof(int), hipMemcpyDeviceToDevice);
		if (value[1] < (n - 1)) hipMemcpy(&cols[n - 1][value[1]], &cols[n][value[1] + 1], (n - value[1] - 1) * sizeof(int), hipMemcpyDeviceToDevice);

		global_matrix_trunc <<< blocks2, threads2 >>>(queue, qsize, lbound, gamma, slice, matrix[n-1], matrix[n], rows[n], cols[n], from, to, m, n, rank);

		n--;
	}
	n--;

the_end:
	/* Освобождаем ранее выделенные ресурсы */

	free(buffer);
	for (int i = 1; i <= n; i++) hipFree(matrix[i]);
	for (int i = 1; i <= n; i++) hipFree(rows[i]);
	for (int i = 1; i <= n; i++) hipFree(cols[i]);
	free(matrix);
	free(rows);
	free(cols);
	hipFree(gamma);
	hipFree(lbound);
	hipFree(queue);
	hipFree(qsize);
	hipFree(from);
	hipFree(to);
	hipFree(slice);
	hipFree(m);

	err = err;
}

int main(int argc, char* argv[])
{
	printf("Title :\t%s\n", title); fflush(stdout);

	if (argc < 3) {
		printf("Usage :\t%s <inputfilename> <outputfilename>\n", argv[0]); fflush(stdout);
		exit(-1);
	}

	char *inputFileName = argv[1];
	char *outputFileName = argv[2];

	char buffer[4096];
	char *tok;
	char *p;
	int n;         /* Ранг текущего массива */
	int *matrix;  /* Стек массивов элементов */
	int i, j;
	int bestPrice;
	int *bestFrom; /* Стек дуг (индексов) в порядке их удаления из матрицы */
	int *bestTo;   /* Стек дуг (индексов) в порядке их удаления из матрицы */

	printf("Input File Name :\t%s\n", inputFileName); fflush(stdout);
	printf("Output File Name :\t%s\n", outputFileName); fflush(stdout);

	FILE *fs = fopen(inputFileName, "r");
	if (fs == NULL) {
		fprintf(stderr, "File open error (%s)\n", inputFileName); fflush(stderr);
		exit(-1);
	}

	n = 0;

	/* Заполняем массив числами из файла */
	/* Операция выполняетя только на хост процессе */
	/* Операция выполняетя в два прохода по файлу */
	/* На первом проходе определяется ранг матрицы */
	/* На втором проходе считываются данные */
	for (i = 0; (tok = fgets(buffer, sizeof(buffer), fs)) != NULL; i++)
	{
		j = 0;
		for (tok = mystrtok(&p, tok, ';'); tok != NULL; tok = mystrtok(&p, NULL, ';'))
		{
			j++;
		}
		n = max(n, j);
	}
	n = max(n, i);

	matrix = (int *)malloc(n*n*sizeof(int));
	bestFrom = (int *)malloc((n + 1)*sizeof(int));
	bestTo = (int *)malloc((n + 1)*sizeof(int));

	fseek(fs, 0, SEEK_SET);

	for (i = 0; (tok = fgets(buffer, sizeof(buffer), fs)) != NULL; i++)
	{
		j = 0;
		for (tok = mystrtok(&p, tok, ';'); tok != NULL; tok = mystrtok(&p, NULL, ';'))
		{
			/* Пустые элементы - это запрещённые пути */
			matrix[n*i + j++] = strempty(tok) ? INT_MAX : atoi(tok);
		}
		for (; j < n; j++) matrix[n*i + j] = INT_MAX;
	}
	for (j = 0; j < (n - i)*n; j++) matrix[n*i + j] = INT_MAX;
	for (i = 0; i < n; i++) matrix[n*i + i] = INT_MAX; /* Запрещаем петли */

	fclose(fs);

	printf("Matrix rank :\t%d\n", n);
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			printf("%d%s", matrix[i*n + j], ((j == n - 1) ? "\n" : "\t"));
		}
	}
	fflush(stdout);

	// Find/set the device.
	int device_qsize = 0;
	hipGetDeviceCount(&device_qsize);
	for (i = 0; i < device_qsize; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		printf("Running on GPU %d (%s)\n", i, properties.name); fflush(stdout);
	}

	host_little(matrix, bestFrom, bestTo, &bestPrice, n);

	hipDeviceReset();

	/* Bыводим результаты */
	if (bestPrice != INT_MAX){
		printf("Best Path\t: "); for (int i = 0; i < n; i++) printf("(%d,%d)%s", bestFrom[i], bestTo[i], ((i < (n - 1)) ? "," : "\n"));
		printf("Best Price\t: %d\n", bestPrice);

		fs = fopen(outputFileName, "w");
		if (fs == NULL) {
			fprintf(stderr, "File open error (%s)\n", outputFileName); fflush(stderr);
			exit(-1);
		}
		for (int i = 0; i < n; i++) fprintf(fs, "%d;%d\n", bestFrom[i], bestTo[i]);
		fclose(fs);
	}


	free(matrix);
	free(bestFrom);
	free(bestTo);

	fflush(stdout);

	if (bestPrice == INT_MAX) exit(-1);
	exit(0);
}