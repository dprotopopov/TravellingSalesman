#include "hip/hip_runtime.h"
﻿char *title = "Floyd's algorithm";
char *description = "Алгоритм Флойда - поиск всех кратчайших путей в графе";
/*
Алгоритм Флойда является одним из методов поиска кратчайших путей в графе. 
В отличии от алгоритма Дейкстры, который позволяет при доведении до конца построить 
ориентированное дерево кратчайших путей от некоторой вершины, метод Флойда позволяет 
найти длины всех кратчайших путей в графе. Конечно эта задача может быть решена 
и многократным применением алгоритма Дейкстры (каждый раз последовательно выбираем 
вершину от первой до N-ной, пока не получим кратчайшие пути от всех вершин графа), 
однако реализация подобной процедуры потребовала бы значительных вычислительных затрат.

Прежде чем представлять алгоритмы, необходимо ввести некоторые обозначения. 
Перенумеруем вершины исходного графа целыми числами от 1 до N. Обозначим через di,jm длину кратчайшего пути 
из вершинм i в вершину j, который в качестве промежуточных может содержать только первые m вершин графа. 
(Напомним, что промежуточной вершиной пути является любая принадлежащая ему вершина, не совпадающая 
с его начальной или конечной вершинами.) Если между вершинами i и j не существует ни одного пути указанного типа, 
то условно будем считать, что di,jm=∞. Из данного определения величин di,jm следует, что величина di,j0, 
представляет длину кратчайшего пути из вершины i в вершину j, не имеющего промежуточных вершин, 
т. е. длину кратчайшей дуги, соединяющей i с j (если такие дуги присутствуют в графе). 
для любой вершины i положим di,im= 0. Отметим далее, что величина di,jmпредставляет длину кратчайшего пути 
между вершинами i и j.

Обозначим через Dm матрицу размера NxN, элемент (i, j) которой совпадает с di,jm. 
Если в исходном графе нам известна длина каждой дуги, то мы можем сформировать матрицу D0. 
Наша цель состоит в определении матрицы DN, представляющей кратчайшие пути между всеми вершинами рассматриваемого графа.

В алгоритме Флойда в качестве исходной выступает матрица D0. 
Вначале из этой матрицы вычисляется матрица D1. 
Затем по матрице D1 вычисляется матрицав D2 и т. д. 
Процесс повторяется до тех пор, пока по матрице DN-1 не будет вычислена матрица DN.

Рассмотрим основную идею, лежащую в основе алгоритма Флойда. 
Суть алгоритма Флойда заключается в проверке того, не окажется ли путь из вершины i в вершину j короче, 
если он будет проходить через некоторую промежуточную вершину m. Предположим, что нам известны:

кратчайший путь из вершины i в вершину m, в котором в качестве промежуточных допускается использование только первых (m - 1) вершин;
кратчайший путь из вершины m в вершину j, в котором в качестве промежуточных допускается использование только первых (m - 1) вершин;
кратчайший путь из вершины i в вершину j, в котором в качестве промежуточных допускается использование только первых (m - 1) вершин.

Поскольку по предположению исходный граф не может содержать контуров отрицательной длины, 
один из двух путей — путь, совпадающий с представленным в пункте 3, или путь, являющийся объединением 
путей из пунктов 1 и 2 — должен быть кратчайшим путем из вершины i в вершину j, 
в котором в качестве промежуточных допускается использование только первых m вершин. Таким образом,

di,jm=min{ di,mm-1+ dm,jm-1; di,jm-1}

Из соотношения видно, что для вычисления элементов матрицы Dm необходимо располагать лишь элементами матрицы Dm-1. 
Более того, соответствующие вычисления могут быть проведены без обращения к исходному графу. 
Теперь мм в состоянии дать формальное описание алгоритма Флойда для нахождения на графе кратчайших путей 
между всеми парами вершин. 

Алгоритм

Перенумеровать вершины графа от 1 до N целыми числами, определить матрицу D0, каждый элемент di,j  
которой есть длина кратчайшей дуги между вершинами i и j. Если такой дуги нет, положить значение элемента 
равным ∞. Кроме того, положить значения диагонального элемента di,iравным 0.
Для целого m, последовательно принимающего значения 1...N определить по элементам матрицы Dm-1 элементы Dm
Алгоритм заканчивается получением матрицы всех кратчайших путей DN, N – число вершин графа.
 
Напомним, для определения по известным элементам матрицы Dm-1 элементов матрицы  Dm 
в алгоритме Флойда применяется рекурсивное соотношение:

di,jm=min{ di,mm-1+ dm,jm-1; di,jm-1}

di,jm – элемент матрицы Dm, di,jm-1 – элементы матрицы Dm-1 найденой на предыдущем шаге алгоритма.
*/
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <ctype.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define assert( bool ) 
int strempty(const char *p)
{
	if (!p)
		return (1);
	for (; *p; p++)
		if (!isspace(*p))
			return (0);
	return (1);
}
char *mystrtok(char **m, char *s, char c)
{
	char *p = s ? s : *m;
	if (!*p)
		return 0;
	*m = strchr(p, c);
	if (*m)
		*(*m)++ = 0;
	else
		*m = p + strlen(p);
	return p;
}
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

__global__ void global_init(long *prev, long *next, int *intermedian, int m, int n){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		intermedian[id] = -1;
	}
}

__global__ void global_floyd(long *prev, long *next, int *intermedian, int m, int n){
	for (int id = blockDim.x*blockIdx.x + threadIdx.x; id < n*n; id += blockDim.x*gridDim.x) {
		int i = id/n;
		int j = id%n;
		if(i != j && prev[i*n+m] != LONG_MAX && prev[m*n+j] != LONG_MAX && (prev[i*n+m] + prev[m*n+j]) < prev[i*n+j]){
			next[id] = prev[i*n+m] + prev[m*n+j];
			intermedian[id] = m;
		}
		else {
			next[id] = prev[id];
		}
	}
}

__host__ void host_floyd(int gridSize, int blockSize, long *matrix, int *intermedian, int n)
{
	hipError_t err;
	long *device_prev;
	long *device_next;
	int *device_intermedian;

	err = hipMalloc((void**)&device_prev, n*n * sizeof(long));
	err = hipMalloc((void**)&device_next, n*n * sizeof(long));
	err = hipMalloc((void**)&device_intermedian, n*n * sizeof(int));

	err = hipMemcpy(device_prev, matrix, n*n*sizeof(long), hipMemcpyHostToDevice);

	int blocks = (gridSize > 0)? gridSize : min(max(1, (int)pow((double)n*n, 0.333333333333333)), 15);
	int threads = (blockSize > 0)? blockSize : min(max(1, (int)pow((double)n*n, 0.333333333333333)), 15);
	global_init <<< blocks, threads >>>(device_prev, device_next, device_intermedian, -1, n);

	for(int m = 0; m < n ; m++){
		global_floyd <<< blocks, threads >>>(device_prev, device_next, device_intermedian, m, n);
		err = hipMemcpy(matrix, device_next, n*n*sizeof(long), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++){
			for (int j = 0; j < n; j++){
				printf("%ld%s", matrix[i*n + j], ((j == n - 1) ? "\n" : ";"));
			}
		}
		printf("\n");
		long * t = device_prev; device_prev = device_next; device_next = t;
	}

	err = hipMemcpy(matrix, device_prev, n*n*sizeof(long), hipMemcpyDeviceToHost);
	err = hipMemcpy(intermedian, device_intermedian, n*n*sizeof(int), hipMemcpyDeviceToHost);

	err = hipFree(device_prev);
	err = hipFree(device_next);
	err = hipFree(device_intermedian);

	err = err;
}

int main(int argc, char* argv[])
{
	int gridSize = 0;
	int blockSize = 0;

	printf("Title :\t%s\n", title); fflush(stdout);

	if (argc < 4) {
		printf("Usage :\t%s [-g <gridSize>] [-b <blockSize>] <inputfilename> <outputfilename> <intermedianfilename>\n", argv[0]); fflush(stdout);
		printf("\tinputfilename - source matrix of path prices or empty\n"); fflush(stdout);
		printf("\toutputfilename - output floyd's matrix of path prices\n"); fflush(stdout);
		printf("\tintermedianfilename - output matrix of intermedian points or empty\n"); fflush(stdout);
		exit(-1);
	}

	int argId = 1;
	for(; argId < argc && argv[argId][0]=='-' ; argId++){
		switch(argv[argId][1]){
		case 'g':
			gridSize = atoi(argv[++argId]);
			break;
		case 'b':
			blockSize = atoi(argv[++argId]);
			break;
		}
	}

	char *inputFileName = argv[argId++];
	char *outputFileName = argv[argId++];
	char *intermedianFileName = argv[argId++];

	printf("Input File Name :\t%s\n", inputFileName); fflush(stdout);
	printf("Output File Name :\t%s\n", outputFileName); fflush(stdout);
	printf("Intermedian File Name :\t%s\n", intermedianFileName); fflush(stdout);

	char buffer[4096];
	char *tok;
	char *p;
	int n;         /* Ранг текущего массива */
	long *matrix;  /* Массив цен */
	int *intermedian;  /* Массивов промежуточных точек */
	int i, j;

	FILE *fs = fopen(inputFileName, "r");
	if (fs == NULL) {
		fprintf(stderr, "File open error (%s)\n", inputFileName); fflush(stderr);
		exit(-1);
	}

	n = 0;

	/* Заполняем массив числами из файла */
	/* Операция выполняетя только на хост процессе */
	/* Операция выполняетя в два прохода по файлу */
	/* На первом проходе определяется ранг матрицы */
	/* На втором проходе считываются данные */
	for (i = 0; (tok = fgets(buffer, sizeof(buffer), fs)) != NULL; i++)
	{
		j = 0;
		for (tok = mystrtok(&p, tok, ';'); tok != NULL; tok = mystrtok(&p, NULL, ';'))
		{
			j++;
		}
		n = max(n, j);
	}
	n = max(n, i);

	matrix = (long *)malloc(n*n*sizeof(long));
	intermedian = (int *)malloc(n*n*sizeof(int));

	fseek(fs, 0, SEEK_SET);

	for (i = 0; (tok = fgets(buffer, sizeof(buffer), fs)) != NULL; i++)
	{
		j = 0;
		for (tok = mystrtok(&p, tok, ';'); tok != NULL; tok = mystrtok(&p, NULL, ';'))
		{
			/* Пустые элементы - это запрещённые пути */
			matrix[n*i + j++] = strempty(tok) ? LONG_MAX : atol(tok);
		}
		for (; j < n; j++) matrix[n*i + j] = LONG_MAX;
	}
	for (j = 0; j < (n - i)*n; j++) matrix[n*i + j] = LONG_MAX;
	for (i = 0; i < n; i++) matrix[n*i + i] = LONG_MAX; /* Запрещаем петли */

	fclose(fs);

	printf("Matrix rank :\t%d\n", n);
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			printf("%ld%s", matrix[i*n + j], ((j == n - 1) ? "\n" : "\t"));
		}
	}
	fflush(stdout);

	// Find/set the device.
	int device_size = 0;
	hipGetDeviceCount(&device_size);
	for (i = 0; i < device_size; ++i)
	{
		hipDeviceProp_t hipDeviceProp_t;
		hipGetDeviceProperties(&hipDeviceProp_t, i);
		printf("Running on GPU %d (%s)\n", i, hipDeviceProp_t.name); 

		printf("Device has ECC support enabled %d\n",hipDeviceProp_t.ECCEnabled);
		printf("Number of asynchronous engines %d\n",hipDeviceProp_t.asyncEngineCount);
		printf("Device can map host memory with hipHostAlloc/hipHostGetDevicePointer %d\n",hipDeviceProp_t.canMapHostMemory);
		printf("Clock frequency in kilohertz %d\n",hipDeviceProp_t.clockRate);
		printf("Compute mode (See hipComputeMode) %d\n",hipDeviceProp_t.computeMode);
		printf("Device can possibly execute multiple kernels concurrently %d\n",hipDeviceProp_t.concurrentKernels);
		printf("Device can concurrently copy memory and execute a kernel. Deprecated. Use instead asyncEngineCount. %d\n",hipDeviceProp_t.deviceOverlap);
		printf("Device is integrated as opposed to discrete %d\n",hipDeviceProp_t.integrated);
		printf("Specified whether there is a run time limit on kernels %d\n",hipDeviceProp_t.kernelExecTimeoutEnabled);
		printf("Size of L2 cache in bytes %d\n",hipDeviceProp_t.l2CacheSize);
		printf("Major compute capability %d\n",hipDeviceProp_t.major);
		printf("Maximum size of each dimension of a grid %d\n",hipDeviceProp_t.maxGridSize[0]);
		printf("Maximum size of each dimension of a grid %d\n",hipDeviceProp_t.maxGridSize[1]);
		printf("Maximum size of each dimension of a grid %d\n",hipDeviceProp_t.maxGridSize[2]);
		printf("Maximum 1D surface size %d\n",hipDeviceProp_t.maxSurface1D);
		printf("Maximum 1D layered surface dimensions %d\n",hipDeviceProp_t.maxSurface1DLayered[0]);
		printf("Maximum 1D layered surface dimensions %d\n",hipDeviceProp_t.maxSurface1DLayered[1]);
		printf("Maximum 2D surface dimensions %d\n",hipDeviceProp_t.maxSurface2D[0]);
		printf("Maximum 2D surface dimensions %d\n",hipDeviceProp_t.maxSurface2D[1]);
		printf("Maximum 2D layered surface dimensions %d\n",hipDeviceProp_t.maxSurface2DLayered[0]);
		printf("Maximum 2D layered surface dimensions %d\n",hipDeviceProp_t.maxSurface2DLayered[1]);
		printf("Maximum 2D layered surface dimensions %d\n",hipDeviceProp_t.maxSurface2DLayered[2]);
		printf("Maximum 3D surface dimensions %d\n",hipDeviceProp_t.maxSurface3D[0]);
		printf("Maximum 3D surface dimensions %d\n",hipDeviceProp_t.maxSurface3D[1]);
		printf("Maximum 3D surface dimensions %d\n",hipDeviceProp_t.maxSurface3D[2]);
		printf("Maximum Cubemap surface dimensions %d\n",hipDeviceProp_t.maxSurfaceCubemap);
		printf("Maximum Cubemap layered surface dimensions %d\n",hipDeviceProp_t.maxSurfaceCubemapLayered[0]);
		printf("Maximum Cubemap layered surface dimensions %d\n",hipDeviceProp_t.maxSurfaceCubemapLayered[1]);
		printf("Maximum 1D texture size %d\n",hipDeviceProp_t.maxTexture1D);
		printf("Maximum 1D layered texture dimensions %d\n",hipDeviceProp_t.maxTexture1DLayered[0]);
		printf("Maximum 1D layered texture dimensions %d\n",hipDeviceProp_t.maxTexture1DLayered[1]);
		printf("Maximum size for 1D textures bound to linear memory %d\n",hipDeviceProp_t.maxTexture1DLinear);
		printf("Maximum 1D mipmapped texture size %d\n",hipDeviceProp_t.maxTexture1DMipmap);
		printf("Maximum 2D texture dimensions %d\n",hipDeviceProp_t.maxTexture2D[0]);
		printf("Maximum 2D texture dimensions %d\n",hipDeviceProp_t.maxTexture2D[1]);
		printf("Maximum 2D texture dimensions if texture gather operations have to be performed %d\n",hipDeviceProp_t.maxTexture2DGather[0]);
		printf("Maximum 2D texture dimensions if texture gather operations have to be performed %d\n",hipDeviceProp_t.maxTexture2DGather[1]);
		printf("Maximum 2D layered texture dimensions %d\n",hipDeviceProp_t.maxTexture2DLayered[0]);
		printf("Maximum 2D layered texture dimensions %d\n",hipDeviceProp_t.maxTexture2DLayered[1]);
		printf("Maximum 2D layered texture dimensions %d\n",hipDeviceProp_t.maxTexture2DLayered[2]);
		printf("Maximum dimensions (width, height, pitch) for 2D textures bound to pitched memory %d\n",hipDeviceProp_t.maxTexture2DLinear[0]);
		printf("Maximum dimensions (width, height, pitch) for 2D textures bound to pitched memory %d\n",hipDeviceProp_t.maxTexture2DLinear[1]);
		printf("Maximum dimensions (width, height, pitch) for 2D textures bound to pitched memory %d\n",hipDeviceProp_t.maxTexture2DLinear[2]);
		printf("Maximum 2D mipmapped texture dimensions %d\n",hipDeviceProp_t.maxTexture2DMipmap[0]);
		printf("Maximum 2D mipmapped texture dimensions %d\n",hipDeviceProp_t.maxTexture2DMipmap[1]);
		printf("Maximum 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3D[0]);
		printf("Maximum 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3D[1]);
		printf("Maximum 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3D[2]);
		printf("Maximum alternate 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3DAlt[0]);
		printf("Maximum alternate 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3DAlt[1]);
		printf("Maximum alternate 3D texture dimensions %d\n",hipDeviceProp_t.maxTexture3DAlt[2]);
		printf("Maximum Cubemap texture dimensions %d\n",hipDeviceProp_t.maxTextureCubemap);
		printf("Maximum Cubemap layered texture dimensions %d\n",hipDeviceProp_t.maxTextureCubemapLayered[0]);
		printf("Maximum Cubemap layered texture dimensions %d\n",hipDeviceProp_t.maxTextureCubemapLayered[1]);
		printf("Maximum size of each dimension of a block %d\n",hipDeviceProp_t.maxThreadsDim[0]);
		printf("Maximum size of each dimension of a block %d\n",hipDeviceProp_t.maxThreadsDim[1]);
		printf("Maximum size of each dimension of a block %d\n",hipDeviceProp_t.maxThreadsDim[2]);
		printf("Maximum number of threads per block %d\n",hipDeviceProp_t.maxThreadsPerBlock);
		printf("Maximum resident threads per multiprocessor %d\n",hipDeviceProp_t.maxThreadsPerMultiProcessor);
		printf("Maximum pitch in bytes allowed by memory copies %d\n",hipDeviceProp_t.memPitch);
		printf("Global memory bus width in bits %d\n",hipDeviceProp_t.memoryBusWidth);
		printf("Peak memory clock frequency in kilohertz %d\n",hipDeviceProp_t.memoryClockRate);
		printf("Minor compute capability %d\n",hipDeviceProp_t.minor);
		printf("Number of multiprocessors on device %d\n",hipDeviceProp_t.multiProcessorCount);
		printf("PCI bus ID of the device %d\n",hipDeviceProp_t.pciBusID);
		printf("PCI device ID of the device %d\n",hipDeviceProp_t.pciDeviceID);
		printf("PCI domain ID of the device %d\n",hipDeviceProp_t.pciDomainID);
		printf("32-bit registers available per block %d\n",hipDeviceProp_t.regsPerBlock);
		printf("Shared memory available per block in bytes %d\n",hipDeviceProp_t.sharedMemPerBlock);
		printf("Device supports stream priorities %d\n",hipDeviceProp_t.streamPrioritiesSupported);
		printf("Alignment requirements for surfaces %d\n",hipDeviceProp_t.surfaceAlignment);
		printf("1 if device is a Tesla device using TCC driver, 0 otherwise %d\n",hipDeviceProp_t.tccDriver);
		printf("Alignment requirement for textures %d\n",hipDeviceProp_t.textureAlignment);
		printf("Pitch alignment requirement for texture references bound to pitched memory %d\n",hipDeviceProp_t.texturePitchAlignment);
		printf("Constant memory available on device in bytes %d\n",hipDeviceProp_t.totalConstMem);
		printf("Global memory available on device in bytes %d\n",hipDeviceProp_t.totalGlobalMem);
		printf("Device shares a unified address space with the host %d\n",hipDeviceProp_t.unifiedAddressing);
		printf("Warp size in threads %d\n",hipDeviceProp_t.warpSize);

		fflush(stdout);
	}

	host_floyd(gridSize, blockSize, matrix, intermedian, n);

	hipDeviceReset();

	/* Bыводим результаты */
	fs = fopen(outputFileName, "w");
	if (fs == NULL) {
		fprintf(stderr, "File open error (%s)\n", outputFileName); fflush(stderr);
		exit(-1);
	}
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			if (matrix[i*n + j] != LONG_MAX)
				fprintf(fs, "%ld%s", matrix[i*n + j], ((j == n - 1) ? "\n" : ";"));
			else
				fprintf(fs, "%s", ((j == n - 1) ? "\n" : ";"));
		}
	}
	fclose(fs);

	/* Bыводим результаты */
	fs = fopen(intermedianFileName, "w");
	if (fs == NULL) {
		fprintf(stderr, "File open error (%s)\n", intermedianFileName); fflush(stderr);
		exit(-1);
	}
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			if (intermedian[i*n + j] >= 0)
				fprintf(fs, "%d%s", intermedian[i*n + j], ((j == n - 1) ? "\n" : ";"));
			else
				fprintf(fs, "%s", ((j == n - 1) ? "\n" : ";"));
		}
	}
	fclose(fs);

	free(matrix);
	free(intermedian);

	exit(0);
}